#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <string.h>
#include <iostream>
#include <stdint.h>
#include <assert.h>
#include <limits.h>
#include <chrono>
#include <stdlib.h> // Para strtoull

#include "json.hpp"

using json = nlohmann::json;
// ***********************************************************************************
// CONSTANTES GLOBALES (DEBEN ESTAR AL PRINCIPIO PARA SER VISIBLES)
// ***********************************************************************************
// Máxima longitud para la cadena concatenada (cadena_base + número_string)
// Asumiendo cadena_base max 200 caracteres, número max 10^18 (19 caracteres), más null terminator
#define MAX_CONCAT_LEN 256
// Máxima longitud para el string del número (ej., 10^18 es 19 dígitos)
#define MAX_NUMBER_STR_LEN 20

// ***********************************************************************************
// MACROS MD5 (DEBEN ESTAR AL PRINCIPIO)
// ***********************************************************************************
// MD5 basic functions.
#define F(x, y, z) (((x) & (y)) | ((~x) & (z)))
#define G(x, y, z) (((x) & (z)) | ((y) & (~z)))
#define H(x, y, z) ((x) ^ (y) ^ (z))
#define I(x, y, z) ((y) ^ ((x) | (~z)))

// Rotate Left function.
#define ROTL32(x, n) (((x) << (n)) | ((x) >> (32 - (n))))

// ***********************************************************************************
// FUNCIONES __DEVICE__ (DEBEN ESTAR DEFINIDAS ANTES DEL KERNEL QUE LAS USA)
// ***********************************************************************************

// MD5 transform function.
// All values are uint32_t.
__device__ void md5_transform(uint32_t *state, const uint32_t *block) {
    uint32_t a = state[0], b = state[1], c = state[2], d = state[3];

    // Round 1
    a = b + ROTL32(a + F(b, c, d) + block[0] + 0xD76AA478, 7);
    d = a + ROTL32(d + F(a, b, c) + block[1] + 0xE8C7B756, 12);
    c = d + ROTL32(c + F(d, a, b) + block[2] + 0x242070DB, 17);
    b = c + ROTL32(b + F(c, d, a) + block[3] + 0xC1BDCEEE, 22);
    a = b + ROTL32(a + F(b, c, d) + block[4] + 0xF57C0FAF, 7);
    d = a + ROTL32(d + F(a, b, c) + block[5] + 0x4787C62A, 12);
    c = d + ROTL32(c + F(d, a, b) + block[6] + 0xA8304613, 17);
    b = c + ROTL32(b + F(c, d, a) + block[7] + 0xFD469501, 22);
    a = b + ROTL32(a + F(b, c, d) + block[8] + 0x698098D8, 7);
    d = a + ROTL32(d + F(a, b, c) + block[9] + 0x8B44F7AF, 12);
    c = d + ROTL32(c + F(d, a, b) + block[10] + 0xFFFF5BB1, 17);
    b = c + ROTL32(b + F(c, d, a) + block[11] + 0x895CD7BE, 22);
    a = b + ROTL32(a + F(b, c, d) + block[12] + 0x6B901122, 7);
    d = a + ROTL32(d + F(a, b, c) + block[13] + 0xFD987193, 12);
    c = d + ROTL32(c + F(d, a, b) + block[14] + 0xA679438E, 17);
    b = c + ROTL32(b + F(c, d, a) + block[15] + 0x49B40821, 22);

    // Round 2
    a = b + ROTL32(a + G(b, c, d) + block[1] + 0xF61E2562, 5);
    d = a + ROTL32(d + G(a, b, c) + block[6] + 0xC040B340, 9);
    c = d + ROTL32(c + G(d, a, b) + block[11] + 0x265E5A51, 14);
    b = c + ROTL32(b + G(c, d, a) + block[0] + 0xE9B6C7AA, 20);
    a = b + ROTL32(a + G(b, c, d) + block[5] + 0xD62F105D, 5);
    d = a + ROTL32(d + G(a, b, c) + block[10] + 0x02441453, 9);
    c = d + ROTL32(c + G(d, a, b) + block[15] + 0xD8A1E681, 14);
    b = c + ROTL32(b + G(c, d, a) + block[4] + 0xE7D3FBC8, 20);
    a = b + ROTL32(a + G(b, c, d) + block[9] + 0x21E1CDE6, 5);
    d = a + ROTL32(d + G(a, b, c) + block[14] + 0xC33707D6, 9);
    c = d + ROTL32(c + G(d, a, b) + block[3] + 0xF4D50D87, 14);
    b = c + ROTL32(b + G(c, d, a) + block[8] + 0x455A14ED, 20);
    a = b + ROTL32(a + G(b, c, d) + block[13] + 0xA9E3E905, 5);
    d = a + ROTL32(d + G(a, b, c) + block[2] + 0xFCEFA3F8, 9);
    c = d + ROTL32(c + G(d, a, b) + block[7] + 0x676F02D9, 14);
    b = c + ROTL32(b + G(c, d, a) + block[12] + 0x8D2A4C8A, 20);

    // Round 3
    a = b + ROTL32(a + H(b, c, d) + block[5] + 0xFFFA3942, 4);
    d = a + ROTL32(d + H(a, b, c) + block[8] + 0x8771F681, 11);
    c = d + ROTL32(c + H(d, a, b) + block[11] + 0x6D9D6122, 16);
    b = c + ROTL32(b + H(c, d, a) + block[14] + 0xFDE5380C, 23);
    a = b + ROTL32(a + H(b, c, d) + block[1] + 0xA4BEEA44, 4);
    d = a + ROTL32(d + H(a, b, c) + block[4] + 0x4BDECFA9, 11);
    c = d + ROTL32(c + H(d, a, b) + block[7] + 0xF6BB4B60, 16);
    b = c + ROTL32(b + H(c, d, a) + block[10] + 0xBEBFBC70, 23);
    a = b + ROTL32(a + H(b, c, d) + block[13] + 0x289B7EC6, 4);
    d = a + ROTL32(d + H(a, b, c) + block[0] + 0xEAA127FA, 11);
    c = d + ROTL32(c + H(d, a, b) + block[3] + 0xFE2CE6E0, 16);
    b = c + ROTL32(b + H(c, d, a) + block[6] + 0xA3014314, 23);
    a = b + ROTL32(a + H(b, c, d) + block[9] + 0x4E0811A1, 4);
    d = a + ROTL32(d + H(a, b, c) + block[12] + 0xF7537E82, 11);
    c = d + ROTL32(c + H(d, a, b) + block[15] + 0xBD3AF235, 16);
    b = c + ROTL32(b + H(c, d, a) + block[2] + 0x2AD7D2BB, 23);

    // Round 4
    a = b + ROTL32(a + I(b, c, d) + block[0] + 0xFEBC46AA, 6);
    d = a + ROTL32(d + I(a, b, c) + block[7] + 0xECD84E7B, 10);
    c = d + ROTL32(c + I(d, a, b) + block[14] + 0x242070DB, 15);
    b = c + ROTL32(b + I(c, d, a) + block[5] + 0x858457D, 21);
    a = b + ROTL32(a + I(b, c, d) + block[12] + 0x6FA87E4F, 6);
    d = a + ROTL32(d + I(a, b, c) + block[3] + 0xFE2CE6E0, 10);
    c = d + ROTL32(c + I(d, a, b) + block[10] + 0xA3014314, 15);
    b = c + ROTL32(b + I(c, d, a) + block[1] + 0x49B40821, 21);
    a = b + ROTL32(a + I(b, c, d) + block[8] + 0x8771F681, 6);
    d = a + ROTL32(d + I(a, b, c) + block[15] + 0xBD3AF235, 10);
    c = d + ROTL32(c + I(d, a, b) + block[6] + 0xF6BB4B60, 15);
    b = c + ROTL32(b + I(c, d, a) + block[13] + 0x289B7EC6, 21);
    a = b + ROTL32(a + I(b, c, d) + block[4] + 0x4BDECFA9, 6);
    d = a + ROTL32(d + I(a, b, c) + block[11] + 0x6D9D6122, 10);
    c = d + ROTL32(c + I(d, a, b) + block[2] + 0x2AD7D2BB, 15);
    b = c + ROTL32(b + I(c, d, a) + block[9] + 0xA9E3E905, 21);

    state[0] += a;
    state[1] += b;
    state[2] += c;
    state[3] += d;
}

// Function to calculate MD5 hash on GPU (adapted from original md5_kernel)
// This will be called from within the main cracking kernel.
__device__ void calculate_md5_hash_on_device(const unsigned char *input_data, unsigned long long input_len, unsigned char *output_hash) {
    uint32_t state[4] = {0x67452301, 0xEFCDAB89, 0x98BADCFE, 0x10325476};

    unsigned long long total_bits = input_len * 8;
    unsigned long long padded_length_bits = total_bits + 1;
    while ((padded_length_bits % 512) != 448) {
        padded_length_bits++;
    }
    padded_length_bits += 64;
    unsigned long long padded_length_bytes = padded_length_bits / 8;
    unsigned long long num_blocks = padded_length_bytes / 64;

    uint32_t current_block[16];

    for (unsigned long long i = 0; i < num_blocks; ++i) {
        for (int j = 0; j < 16; ++j) {
            unsigned long long byte_idx = i * 64 + j * 4;
            current_block[j] = 0;

            for (int k = 0; k < 4; ++k) {
                if (byte_idx + k < input_len) {
                    current_block[j] |= ((uint32_t)input_data[byte_idx + k]) << (k * 8);
                } else if (byte_idx + k == input_len) {
                    current_block[j] |= ((uint32_t)0x80) << (k * 8);
                }
            }
        }

        if (i == num_blocks - 1) {
            current_block[14] = (uint32_t)(total_bits & 0xFFFFFFFF);
            current_block[15] = (uint32_t)(total_bits >> 32);
        }

        md5_transform(state, current_block);
    }

    for (int i = 0; i < 4; ++i) {
        output_hash[i * 4 + 0] = (unsigned char)(state[i] & 0xFF);
        output_hash[i * 4 + 1] = (unsigned char)((state[i] >> 8) & 0xFF);
        output_hash[i * 4 + 2] = (unsigned char)((state[i] >> 16) & 0xFF);
        output_hash[i * 4 + 3] = (unsigned char)((state[i] >> 24) & 0xFF);
    }
}

// Helper function to convert unsigned long long to string on device
// Returns length of the string
__device__ int ulltoa_device(unsigned long long value, char* buffer) {
    if (value == 0) {
        buffer[0] = '0';
        buffer[1] = '\0';
        return 1;
    }
    int i = 0;
    char temp_buffer[20]; // Max for ULL is 20 digits + null terminator
    int j = 0;
    unsigned long long temp_val = value; // Use a temp variable for calculation
    while (temp_val > 0) {
        temp_buffer[j++] = (temp_val % 10) + '0';
        temp_val /= 10;
    }
    // Reverse the string
    while (j > 0) {
        buffer[i++] = temp_buffer[--j];
    }
    buffer[i] = '\0';
    return i;
}

// Helper function to concatenate strings on device
__device__ int concatenate_device(char* dest, const char* s1, int len1, const char* s2, int len2) {
    for (int i = 0; i < len1; ++i) {
        dest[i] = s1[i];
    }
    for (int i = 0; i < len2; ++i) {
        dest[len1 + i] = s2[i];
    }
    dest[len1 + len2] = '\0';
    return len1 + len2;
}

// ***********************************************************************************
// KERNEL __GLOBAL__ (DEBE ESTAR DEFINIDO ANTES DE MAIN)
// ***********************************************************************************
__global__ void md5_prefix_cracker_kernel(
    const unsigned char* d_block_base_string, unsigned long long block_base_string_len, // Nueva cadena base
    const unsigned char* d_target_prefix_bytes, unsigned int target_prefix_len,
    volatile int* d_found_flag,
    unsigned char* d_found_hash,
    char* d_found_number_string,
    unsigned long long global_start_range,  //Inicio del rango de búsqueda global
    unsigned long long global_end_range     //Fin del rango de búsqueda global
)

{
    unsigned long long thread_id = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned long long num_threads_in_grid = gridDim.x * blockDim.x;

    // Calcular el total de números en el rango global
    // Evitar desbordamiento si global_end_range es ULLONG_MAX y global_start_range es muy pequeño
    unsigned long long total_numbers_to_search;
    if (global_end_range < global_start_range) { // Rango inválido o vacío
        total_numbers_to_search = 0;
    } else {
        total_numbers_to_search = global_end_range - global_start_range + 1;
    }

    // Si el rango es inválido o no hay números, el hilo termina
    if (total_numbers_to_search == 0) {
        return;
    }

    // Distribuir el rango de números equitativamente entre los hilos
    unsigned long long numbers_per_thread_segment_base = total_numbers_to_search / num_threads_in_grid;
    unsigned long long remainder = total_numbers_to_search % num_threads_in_grid;

    // Calcular el inicio del segmento para este hilo
    unsigned long long thread_start_offset = global_start_range + thread_id * numbers_per_thread_segment_base;
    if (thread_id < remainder) { // Los primeros 'remainder' hilos reciben un número extra
        thread_start_offset += thread_id;
    } else {
        thread_start_offset += remainder; // Los hilos restantes ajustan su offset por los extras ya distribuidos
    }

    // Calcular el fin del segmento para este hilo
    unsigned long long thread_end_offset = thread_start_offset + numbers_per_thread_segment_base - 1;
    if (thread_id < remainder) {
        thread_end_offset++; // Suma el número extra si este hilo lo recibió
    }

    // Asegurarse de que el fin del segmento no exceda el fin del rango global
    if (thread_end_offset > global_end_range) {
        thread_end_offset = global_end_range;
    }

    // Asegurarse de que el inicio no exceda el fin del propio segmento o el fin global
    // Esto maneja casos donde numbers_per_thread_segment_base podría ser 0 o muy pequeño
    if (thread_start_offset > global_end_range || thread_start_offset > thread_end_offset) {
        return; // Segmento inválido o vacío para este hilo
    }


    // Buffers locales para la cadena concatenada y el número convertido a string
    char concatenated_string_buffer[MAX_CONCAT_LEN];
    char number_str_buffer[MAX_NUMBER_STR_LEN];
    unsigned char current_hash[16]; // Para almacenar el hash calculado por este hilo

    // Copiar la cadena base del bloque (hash previo + transacciones serializadas) al buffer local UNA VEZ por hilo
    // Esto es más eficiente que copiarlo en cada iteración del bucle si es estático para la tarea.
    // Asumiendo que block_base_string_len ya incluye el null terminator si viene de strlen.
    // Si no, asegúrate de que el buffer tenga espacio para ello.
    for (int i = 0; i < block_base_string_len; ++i) {
        concatenated_string_buffer[i] = d_block_base_string[i];
    }

    // Bucle principal para que el hilo itere sobre su rango asignado
    for (unsigned long long current_number = thread_start_offset; current_number <= thread_end_offset; ++current_number) {
        // Verificar si una solución ya fue encontrada por otro hilo (lectura volátil para salida temprana)
        if (*d_found_flag == 1) {
             return;
        }

        // Convertir el número actual a string
        int num_str_len = ulltoa_device(current_number, number_str_buffer);

        // Concatenar la cadena base del bloque con el número del nonce
        // La cadena base del bloque ya está en concatenated_string_buffer
        // El nuevo len de la cadena base ahora es block_base_string_len
        int full_string_len = concatenate_device(
            concatenated_string_buffer,
            (const char*)d_block_base_string, (int)block_base_string_len, // Usar d_block_base_string directamente
            number_str_buffer, num_str_len
        );

        // Calcular el hash MD5
        calculate_md5_hash_on_device((const unsigned char*)concatenated_string_buffer, full_string_len, current_hash);

        // Comparar el prefijo del hash calculado con el objetivo
        bool prefix_matches = true;
        for (unsigned int k = 0; k < target_prefix_len; ++k) {
            if (current_hash[k] != d_target_prefix_bytes[k]) {
                prefix_matches = false;
                break;
            }
        }

        // Si el prefijo coincide, actualizar la bandera y almacenar resultados (atómicamente)
        if (prefix_matches) {
            if (atomicCAS((int*)d_found_flag, 0, 1) == 0) {
                for (int k = 0; k < 16; ++k) {
                    d_found_hash[k] = current_hash[k];
                }
                int str_idx = 0;
                while(number_str_buffer[str_idx] != '\0' && str_idx < MAX_NUMBER_STR_LEN) {
                    d_found_number_string[str_idx] = number_str_buffer[str_idx];
                    str_idx++;
                }
                d_found_number_string[str_idx] = '\0';
            }
            return; // El hilo termina después de encontrar o confirmar que ya se encontró
        }
    }
}


// ***********************************************************************************
// FUNCIONES DE AYUDA DEL HOST (PUEDEN IR DESPUÉS DEL KERNEL PERO ANTES DE MAIN)
// ***********************************************************************************

// Función de ayuda para verificar errores de CUDA
#define CHECK_CUDA_ERROR(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess)
   {
      fprintf(stderr,"CUDA Error: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

// Converts a hexadecimal character to its integer value
int hexCharToInt(char c) {
    if (c >= '0' && c <= '9') return c - '0';
    if (c >= 'a' && c <= 'f') return c - 'a' + 10;
    if (c >= 'A' && c <= 'F') return c - 'A' + 10;
    return -1; // Invalid hex char
}

// Converts a hexadecimal string to a byte array
// Returns the number of bytes written to byte_array, or -1 on error
int hexStringToBytes(const char* hex_string, unsigned char* byte_array) {
    int len = strlen(hex_string);
    if (len % 2 != 0) {
        // Hex string must have an even number of characters
        // Or if it's a single hex digit, convert it to 0X style
        if (len == 1) { // Handle "1" becoming "01"
            int nibble = hexCharToInt(hex_string[0]);
            if (nibble == -1) return -1;
            byte_array[0] = (unsigned char)nibble; // Will be 0X
            return 1;
        }
        return -1;
    }
    int byte_len = len / 2;
    for (int i = 0; i < byte_len; ++i) {
        int high_nibble = hexCharToInt(hex_string[i * 2]);
        int low_nibble = hexCharToInt(hex_string[i * 2 + 1]);
        if (high_nibble == -1 || low_nibble == -1) {
            return -1; // Invalid hex character
        }
        byte_array[i] = (high_nibble << 4) | low_nibble;
    }
    return byte_len;
}

// ***********************************************************************************
// FUNCIÓN MAIN (DEBE SER LA ÚLTIMA FUNCIÓN EJECUTABLE)
// ***********************************************************************************
int main(int argc, char* argv[]) {
    // Se espera 1 argumento: la cadena JSON de la tarea
    if (argc != 2) {
        fprintf(stderr, "Uso: %s <tarea_json_string>\n", argv[0]);
        fprintf(stderr, "Ejemplo: %s '{\"prev_hash\": \"000...\", \"transacciones\": \"[{\"de\":\"A\"...}]\", \"dificultad\": \"00\", \"start_nonce\": 0, \"end_nonce\": 1000000}'\n", argv[0]);
        return 1;
    }

    // --- 1. Leer y parsear el JSON de la tarea ---
    std::string task_json_string = argv[1];
    json task_data;
    try {
        task_data = json::parse(task_json_string);
    } catch (const json::parse_error& e) {
        fprintf(stderr, "Error al parsear el JSON de la tarea: %s\n", e.what());
        return 1;
    }

    // Extraer datos de la tarea JSON
    std::string prev_hash_str;
    std::string transactions_str; // Las transacciones como una cadena JSON serializada
    std::string difficulty_prefix_str;
    unsigned long long h_start_number;
    unsigned long long h_end_number;

    try {
        prev_hash_str = task_data.at("prev_hash").get<std::string>();
        transactions_str = task_data.at("transacciones").dump(); // Las transacciones son un array JSON, lo serializamos a string
        difficulty_prefix_str = task_data.at("dificultad").get<std::string>();
        h_start_number = task_data.at("start_nonce").get<unsigned long long>();
        h_end_number = task_data.at("end_nonce").get<unsigned long long>();
    } catch (const json::exception& e) {
        fprintf(stderr, "Error: Falta un campo requerido en el JSON de la tarea o formato incorrecto: %s\n", e.what());
        return 1;
    }

    // Construir la cadena base del bloque para el hash (prev_hash + transacciones_serializadas)
    // El formato del bloque para el hash es: hash_anterior + json_transacciones + nonce
    // Así que la cadena base es: hash_anterior + json_transacciones
    std::string block_base_string_std = prev_hash_str + transactions_str;
    const unsigned char* h_block_base_string = (const unsigned char*)block_base_string_std.c_str();
    unsigned long long block_base_string_len = block_base_string_std.length(); // Usar .length() para std::string

    // Verificar si la cadena base más el nonce excederán el buffer
    if (block_base_string_len + MAX_NUMBER_STR_LEN >= MAX_CONCAT_LEN) {
        fprintf(stderr, "Error: La longitud de la cadena base del bloque (%llu) más el nonce exceden el buffer (%d). Aumente MAX_CONCAT_LEN o revise los datos.\n", block_base_string_len, MAX_CONCAT_LEN);
        return 1;
    }

    unsigned char h_target_prefix_bytes[16]; // Max MD5 hash size
    // Convertir el prefijo de dificultad hexadecimal a bytes
    int target_prefix_len = hexStringToBytes(difficulty_prefix_str.c_str(), h_target_prefix_bytes);

    if (target_prefix_len == -1 || target_prefix_len == 0 || target_prefix_len > 16) {
        fprintf(stderr, "Error: El prefijo de dificultad (%s) es inválido o su longitud no es adecuada (max 32 caracteres hex / 16 bytes). Longitud de bytes: %d\n", difficulty_prefix_str.c_str(), target_prefix_len);
        return 1;
    }

    if (h_start_number > h_end_number) {
        fprintf(stderr, "Error: El número de inicio del nonce (%llu) no puede ser mayor que el número de fin (%llu).\n", h_start_number, h_end_number);
        return 1;
    }
    
    unsigned long long total_numbers_to_search;
    if (h_end_number < h_start_number) { // Debería haber sido manejado por el if anterior, pero como seguridad
        total_numbers_to_search = 0;
    } else {
        total_numbers_to_search = h_end_number - h_start_number + 1;
    }

    if (total_numbers_to_search == 0) {
        // En este caso, el Worker no debería haber enviado una tarea así, pero lo manejamos.
        json result_json;
        result_json["status"] = "no_solution_found";
        result_json["reason"] = "Rango de nonce vacío.";
        std::cout << result_json.dump() << std::endl; // Imprime JSON a stdout
        return 0; // Termina con éxito, pero sin solución
    }


    // --- Variables en el Host (punteros a memoria del Device) ---
    unsigned char* d_block_base_string = nullptr; // Nuevo nombre para claridad
    unsigned char* d_base_string = nullptr;
    unsigned char* d_target_prefix_bytes = nullptr;
    volatile int* d_found_flag = nullptr;
    unsigned char* d_found_hash = nullptr;
    char* d_found_number_string = nullptr;

    // --- Parámetros de Configuración del Kernel ---
    // Estos valores determinan cuántos hilos y bloques se lanzan.
    // El rango de búsqueda ahora lo define el usuario.
    const unsigned int NUM_BLOCKS = 128;
    const unsigned int THREADS_PER_BLOCK = 256;
    const unsigned long long TOTAL_THREADS_IN_GRID = (unsigned long long)NUM_BLOCKS * THREADS_PER_BLOCK;


    // Impresión de estado (para depuración, se puede quitar en producción)
    fprintf(stderr, "Iniciando minado para hash previo: '%s', dificultad: '%s'\n", prev_hash_str.c_str(), difficulty_prefix_str.c_str());
    fprintf(stderr, "Rango de nonce: desde %llu hasta %llu\n", h_start_number, h_end_number);
    fprintf(stderr, "Longitud del prefijo a comparar (en bytes): %d\n", target_prefix_len);
    fprintf(stderr, "Espacio de búsqueda total en este rango: %llu nonces\n", total_numbers_to_search);
    fprintf(stderr, "--- Minando, por favor espere ---\n");


    // 1. Asignar memoria en el device
    CHECK_CUDA_ERROR(hipMalloc((void**)&d_block_base_string, block_base_string_len + 1)); // +1 para el null terminator
    CHECK_CUDA_ERROR(hipMalloc((void**)&d_target_prefix_bytes, target_prefix_len));
    CHECK_CUDA_ERROR(hipMalloc((void**)&d_found_flag, sizeof(int)));
    CHECK_CUDA_ERROR(hipMalloc((void**)&d_found_hash, 16));
    CHECK_CUDA_ERROR(hipMalloc((void**)&d_found_number_string, MAX_NUMBER_STR_LEN)); // Aquí usa la constante


    // 2. Copiar datos del host al device
    CHECK_CUDA_ERROR(hipMemcpy(d_block_base_string, h_block_base_string, block_base_string_len + 1, hipMemcpyHostToDevice));
    CHECK_CUDA_ERROR(hipMemcpy(d_target_prefix_bytes, h_target_prefix_bytes, target_prefix_len, hipMemcpyHostToDevice));
    CHECK_CUDA_ERROR(hipMemset((void*)d_found_flag, 0, sizeof(int)));

    // 3. Configurar y lanzar el kernel de búsqueda
    dim3 blocks(NUM_BLOCKS);
    dim3 threads(THREADS_PER_BLOCK);

    // --- INICIO MEDICIÓN DE TIEMPO ---
    auto start_time = std::chrono::high_resolution_clock::now();

    md5_prefix_cracker_kernel<<<blocks, threads>>>(
        d_block_base_string, block_base_string_len,
        d_target_prefix_bytes, target_prefix_len,
        d_found_flag, d_found_hash, d_found_number_string,
        h_start_number,    // Pasa el inicio del rango
        h_end_number       // Pasa el fin del rango
    );
    CHECK_CUDA_ERROR(hipGetLastError());

    // 4. Sincronizar el dispositivo y finalizar la medición
    CHECK_CUDA_ERROR(hipDeviceSynchronize());
    auto end_time = std::chrono::high_resolution_clock::now();
    // --- FIN MEDICIÓN DE TIEMPO ---

    std::chrono::duration<double, std::milli> duration = end_time - start_time;

    // 5. Verificar si se encontró una solución y copiar resultados
    int h_found_flag = 0;
    CHECK_CUDA_ERROR(hipMemcpy(&h_found_flag, (const void*)d_found_flag, sizeof(int), hipMemcpyDeviceToHost));

    json result_json; // Objeto JSON para la salida

    if (h_found_flag == 1) {
        unsigned char h_final_hash[16];
        char h_final_number_string[MAX_NUMBER_STR_LEN];
        CHECK_CUDA_ERROR(hipMemcpy(h_final_hash, d_found_hash, 16, hipMemcpyDeviceToHost));
        CHECK_CUDA_ERROR(hipMemcpy(h_final_number_string, d_found_number_string, MAX_NUMBER_STR_LEN, hipMemcpyDeviceToHost));

        // Convertir el hash binario a string hexadecimal para la salida JSON
        char final_hash_hex_str[33]; // 16 bytes * 2 chars/byte + null terminator
        for (int i = 0; i < 16; ++i) {
            sprintf(&final_hash_hex_str[i*2], "%02x", (unsigned char)h_final_hash[i]);
        }
        final_hash_hex_str[32] = '\0'; // Asegura el null terminator

        result_json["status"] = "solution_found";
        result_json["nonce_found"] = std::stoull(h_final_number_string); // Convierte string a ULL
        result_json["block_hash_result"] = std::string(final_hash_hex_str);
        result_json["elapsed_time_ms"] = duration.count();

        fprintf(stderr, "\n--- SOLUCIÓN ENCONTRADA ---\n");
        fprintf(stderr, "Nonce: %s\n", h_final_number_string);
        fprintf(stderr, "Hash MD5 resultante: %s\n", final_hash_hex_str);
        fprintf(stderr, "Tiempo de ejecución: %.2f ms\n", duration.count());

    } else {
        result_json["status"] = "no_solution_found";
        result_json["elapsed_time_ms"] = duration.count();
        result_json["reason"] = "No se encontró un nonce en el rango especificado que cumpla con la dificultad.";
        fprintf(stderr, "\n--- No se encontró una solución en el rango [%llu - %llu] ---\n", h_start_number, h_end_number);
        fprintf(stderr, "Tiempo de ejecución: %.2f ms\n", duration.count());
    }

    // 6. Liberar memoria
    hipFree((void*)d_found_flag);
    hipFree(d_block_base_string);
    hipFree(d_target_prefix_bytes);
    hipFree(d_found_hash);
    hipFree(d_found_number_string);

    // 7. Imprimir el resultado JSON a la salida estándar
    std::cout << result_json.dump() << std::endl;

    return 0;
}
