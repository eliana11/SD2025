#include "hip/hip_runtime.h"
#include <iostream>
#include <string>
#include <vector>
#include <chrono>
#include <cstring> // For memcpy, strlen
#include <cstdio>  // For sprintf, fprintf
#include <algorithm> // For std::reverse
#include <stdint.h>
#include <limits.h> // For ULLONG_MAX
#include <stdlib.h> // For strtoull

#include "json.hpp" // Asegúrate de que nlohmann/json.hpp esté en tu include path

// Incluir CUDA solo si se está compilando con nvcc
#ifdef __HIPCC__
#include <hip/hip_runtime.h>
#endif

using json = nlohmann::json;

// --- CONSTANTES GLOBALES (Comunes para CPU y GPU) ---
#define MAX_CONCAT_LEN 256
#define MAX_NUMBER_STR_LEN 20

// --- MACROS MD5 (Comunes para CPU y GPU) ---
#define F(x, y, z) (((x) & (y)) | ((~x) & (z)))
#define G(x, y, z) (((x) & (z)) | ((y) & (~z)))
#define H(x, y, z) ((x) ^ (y) ^ (z))
#define I(x, y, z) ((y) ^ ((x) | (~z)))

#define ROTL32(x, n) (((x) << (n)) | ((x) >> (32 - (n))))

// --- FUNCIONES DE SOPORTE GENERALES (Para Host, usadas por CPU y por la parte Host de GPU) ---

// Converts a hexadecimal character to its integer value
int hexCharToInt(char c) {
    if (c >= '0' && c <= '9') return c - '0';
    if (c >= 'a' && c <= 'f') return c - 'a' + 10;
    if (c >= 'A' && c <= 'F') return c - 'A' + 10;
    return -1; // Invalid hex char
}

// Converts a hexadecimal string to a byte array
// Returns the number of bytes written to byte_array, or -1 on error
int hexStringToBytes(const char* hex_string, unsigned char* byte_array) {
    int len = std::strlen(hex_string);
    if (len % 2 != 0) {
        if (len == 1) { // Handle "1" becoming "01" (e.g., if difficulty is "1" it means "01")
            int nibble = hexCharToInt(hex_string[0]);
            if (nibble == -1) return -1;
            byte_array[0] = (unsigned char)nibble;
            return 1;
        }
        return -1;
    }
    int byte_len = len / 2;
    for (int i = 0; i < byte_len; ++i) {
        int high_nibble = hexCharToInt(hex_string[i * 2]);
        int low_nibble = hexCharToInt(hex_string[i * 2 + 1]);
        if (high_nibble == -1 || low_nibble == -1) {
            return -1; // Invalid hex character
        }
        byte_array[i] = (high_nibble << 4) | low_nibble;
    }
    return byte_len;
}

// Helper function to convert unsigned long long to string for CPU (Host)
// Used by CPU miner and by the Host-side preparation for GPU.
int ulltoa_host(unsigned long long value, char* buffer) {
    if (value == 0) {
        buffer[0] = '0';
        buffer[1] = '\0';
        return 1;
    }
    int i = 0;
    char temp_buffer[MAX_NUMBER_STR_LEN]; // Max for ULL is 20 digits + null terminator
    int j = 0;
    unsigned long long temp_val = value;
    while (temp_val > 0) {
        temp_buffer[j++] = (temp_val % 10) + '0';
        temp_val /= 10;
    }
    // Reverse the string
    while (j > 0) {
        buffer[i++] = temp_buffer[--j];
    }
    buffer[i] = '\0';
    return i;
}

// Helper function to concatenate strings for CPU (Host)
// Used by CPU miner.
int concatenate_host(char* dest, const char* s1, int len1, const char* s2, int len2) {
    if (len1 + len2 >= MAX_CONCAT_LEN) {
        fprintf(stderr, "Error: Buffer de concatenación insuficiente en concatenate_host. (Requested %d, Max %d)\n", len1 + len2, MAX_CONCAT_LEN);
        // Depending on desired error handling, you might exit or return an error code
        return -1; 
    }
    std::memcpy(dest, s1, len1);
    std::memcpy(dest + len1, s2, len2);
    dest[len1 + len2] = '\0';
    return len1 + len2;
}


// --- CPU MINER IMPLEMENTATION ---

// MD5 transform function for CPU.
void md5_transform_cpu(uint32_t *state, const uint32_t *block) {
    uint32_t a = state[0], b = state[1], c = state[2], d = state[3];

    // Round 1
    a = b + ROTL32(a + F(b, c, d) + block[0] + 0xD76AA478, 7);
    d = a + ROTL32(d + F(a, b, c) + block[1] + 0xE8C7B756, 12);
    c = d + ROTL32(c + F(d, a, b) + block[2] + 0x242070DB, 17);
    b = c + ROTL32(b + F(c, d, a) + block[3] + 0xC1BDCEEE, 22);
    a = b + ROTL32(a + F(b, c, d) + block[4] + 0xF57C0FAF, 7);
    d = a + ROTL32(d + F(a, b, c) + block[5] + 0x4787C62A, 12);
    c = d + ROTL32(c + F(d, a, b) + block[6] + 0xA8304613, 17);
    b = c + ROTL32(b + F(c, d, a) + block[7] + 0xFD469501, 22);
    a = b + ROTL32(a + F(b, c, d) + block[8] + 0x698098D8, 7);
    d = a + ROTL32(d + F(a, b, c) + block[9] + 0x8B44F7AF, 12);
    c = d + ROTL32(c + F(d, a, b) + block[10] + 0xFFFF5BB1, 17);
    b = c + ROTL32(b + F(c, d, a) + block[11] + 0x895CD7BE, 22);
    a = b + ROTL32(a + F(b, c, d) + block[12] + 0x6B901122, 7);
    d = a + ROTL32(d + F(a, b, c) + block[13] + 0xFD987193, 12);
    c = d + ROTL32(c + F(d, a, b) + block[14] + 0xA679438E, 17);
    b = c + ROTL32(b + F(c, d, a) + block[15] + 0x49B40821, 22);

    // Round 2
    a = b + ROTL32(a + G(b, c, d) + block[1] + 0xF61E2562, 5);
    d = a + ROTL32(d + G(a, b, c) + block[6] + 0xC040B340, 9);
    c = d + ROTL32(c + G(d, a, b) + block[11] + 0x265E5A51, 14);
    b = c + ROTL32(b + G(c, d, a) + block[0] + 0xE9B6C7AA, 20);
    a = b + ROTL32(a + G(b, c, d) + block[5] + 0xD62F105D, 5);
    d = a + ROTL32(d + G(a, b, c) + block[10] + 0x02441453, 9);
    c = d + ROTL32(c + G(d, a, b) + block[15] + 0xD8A1E681, 14);
    b = c + ROTL32(b + G(c, d, a) + block[4] + 0xE7D3FBC8, 20);
    a = b + ROTL32(a + G(b, c, d) + block[9] + 0x21E1CDE6, 5);
    d = a + ROTL32(d + G(a, b, c) + block[14] + 0xC33707D6, 9);
    c = d + ROTL32(c + G(d, a, b) + block[3] + 0xF4D50D87, 14);
    b = c + ROTL32(b + G(c, d, a) + block[8] + 0x455A14ED, 20);
    a = b + ROTL32(a + G(b, c, d) + block[13] + 0xA9E3E905, 5);
    d = a + ROTL32(d + G(a, b, c) + block[2] + 0xFCEFA3F8, 9);
    c = d + ROTL32(c + G(d, a, b) + block[7] + 0x676F02D9, 14);
    b = c + ROTL32(b + G(c, d, a) + block[12] + 0x8D2A4C8A, 20);

    // Round 3
    a = b + ROTL32(a + H(b, c, d) + block[5] + 0xFFFA3942, 4);
    d = a + ROTL32(d + H(a, b, c) + block[8] + 0x8771F681, 11);
    c = d + ROTL32(c + H(d, a, b) + block[11] + 0x6D9D6122, 16);
    b = c + ROTL32(b + H(c, d, a) + block[14] + 0xFDE5380C, 23);
    a = b + ROTL32(a + H(b, c, d) + block[1] + 0xA4BEEA44, 4);
    d = a + ROTL32(d + H(a, b, c) + block[4] + 0x4BDECFA9, 11);
    c = d + ROTL32(c + H(d, a, b) + block[7] + 0xF6BB4B60, 16);
    b = c + ROTL32(b + H(c, d, a) + block[10] + 0xBEBFBC70, 23);
    a = b + ROTL32(a + H(b, c, d) + block[13] + 0x289B7EC6, 4);
    d = a + ROTL32(d + H(a, b, c) + block[0] + 0xEAA127FA, 11);
    c = d + ROTL32(c + H(d, a, b) + block[3] + 0xFE2CE6E0, 16);
    b = c + ROTL32(b + H(c, d, a) + block[6] + 0xA3014314, 23);
    a = b + ROTL32(a + H(b, c, d) + block[9] + 0x4E0811A1, 4);
    d = a + ROTL32(d + H(a, b, c) + block[12] + 0xF7537E82, 11);
    c = d + ROTL32(c + H(d, a, b) + block[15] + 0xBD3AF235, 16);
    b = c + ROTL32(b + H(c, d, a) + block[2] + 0x2AD7D2BB, 23);

    // Round 4
    a = b + ROTL32(a + I(b, c, d) + block[0] + 0xFEBC46AA, 6);
    d = a + ROTL32(d + I(a, b, c) + block[7] + 0xECD84E7B, 10);
    c = d + ROTL32(c + I(d, a, b) + block[14] + 0x242070DB, 15);
    b = c + ROTL32(b + I(c, d, a) + block[5] + 0x858457D, 21);
    a = b + ROTL32(a + I(b, c, d) + block[12] + 0x6FA87E4F, 6);
    d = a + ROTL32(d + I(a, b, c) + block[3] + 0xFE2CE6E0, 10);
    c = d + ROTL32(c + I(d, a, b) + block[10] + 0xA3014314, 15);
    b = c + ROTL32(b + I(c, d, a) + block[1] + 0x49B40821, 21);
    a = b + ROTL32(a + I(b, c, d) + block[8] + 0x8771F681, 6);
    d = a + ROTL32(d + I(a, b, c) + block[15] + 0xBD3AF235, 10);
    c = d + ROTL32(c + I(d, a, b) + block[6] + 0xF6BB4B60, 15);
    b = c + ROTL32(b + I(c, d, a) + block[13] + 0x289B7EC6, 21);
    a = b + ROTL32(a + I(b, c, d) + block[4] + 0x4BDECFA9, 6);
    d = a + ROTL32(d + I(a, b, c) + block[11] + 0x6D9D6122, 10);
    c = d + ROTL32(c + I(d, a, b) + block[2] + 0x2AD7D2BB, 15);
    b = c + ROTL32(b + I(c, d, a) + block[9] + 0xA9E3E905, 21);

    state[0] += a;
    state[1] += b;
    state[2] += c;
    state[3] += d;
}

// Function to calculate MD5 hash on CPU
void calculate_md5_hash_cpu(const unsigned char *input_data, unsigned long long input_len, unsigned char *output_hash) {
    uint32_t state[4] = {0x67452301, 0xEFCDAB89, 0x98BADCFE, 0x10325476};

    unsigned long long total_bits = input_len * 8;
    unsigned long long padded_length_bits = total_bits + 1;
    while ((padded_length_bits % 512) != 448) {
        padded_length_bits++;
    }
    padded_length_bits += 64;
    unsigned long long padded_length_bytes = padded_length_bits / 8;
    unsigned long long num_blocks = padded_length_bytes / 64;

    uint32_t current_block[16];

    for (unsigned long long i = 0; i < num_blocks; ++i) {
        for (int j = 0; j < 16; ++j) {
            unsigned long long byte_idx = i * 64 + j * 4;
            current_block[j] = 0;

            for (int k = 0; k < 4; ++k) {
                if (byte_idx + k < input_len) {
                    current_block[j] |= ((uint32_t)input_data[byte_idx + k]) << (k * 8);
                } else if (byte_idx + k == input_len) {
                    current_block[j] |= ((uint32_t)0x80) << (k * 8);
                }
            }
        }

        if (i == num_blocks - 1) {
            current_block[14] = (uint32_t)(total_bits & 0xFFFFFFFF);
            current_block[15] = (uint32_t)(total_bits >> 32);
        }

        md5_transform_cpu(state, current_block);
    }

    for (int i = 0; i < 4; ++i) {
        output_hash[i * 4 + 0] = (unsigned char)(state[i] & 0xFF);
        output_hash[i * 4 + 1] = (unsigned char)((state[i] >> 8) & 0xFF);
        output_hash[i * 4 + 2] = (unsigned char)((state[i] >> 16) & 0xFF);
        output_hash[i * 4 + 3] = (unsigned char)((state[i] >> 24) & 0xFF);
    }
}

// Main cracking function for CPU (single-threaded)
bool md5_prefix_cracker_cpu_wrapper(
    const unsigned char* block_base_string, unsigned long long block_base_string_len,
    const unsigned char* target_prefix_bytes, unsigned int target_prefix_len,
    unsigned char* found_hash, char* found_number_string,
    unsigned long long global_start_range, unsigned long long global_end_range
) {
    for (unsigned long long current_number = global_start_range; current_number <= global_end_range; ++current_number) {
        char concatenated_string_buffer[MAX_CONCAT_LEN];
        char number_str_buffer[MAX_NUMBER_STR_LEN];
        unsigned char current_hash[16];

        int num_str_len = ulltoa_host(current_number, number_str_buffer);
        if (num_str_len == -1) return false; // Error converting number to string

        int full_string_len = concatenate_host(
            concatenated_string_buffer,
            (const char*)block_base_string, (int)block_base_string_len,
            number_str_buffer, num_str_len
        );
        if (full_string_len == -1) return false; // Error concatenating strings

        calculate_md5_hash_cpu((const unsigned char*)concatenated_string_buffer, full_string_len, current_hash);

        bool prefix_matches = true;
        for (unsigned int k = 0; k < target_prefix_len; ++k) {
            if (current_hash[k] != target_prefix_bytes[k]) {
                prefix_matches = false;
                break;
            }
        }

        if (prefix_matches) {
            std::memcpy(found_hash, current_hash, 16);
            std::memcpy(found_number_string, number_str_buffer, num_str_len + 1);
            return true;
        }
    }
    return false;
}

// --- GPU MINER IMPLEMENTATION (only compiled if __HIPCC__ is defined) ---

#ifdef __HIPCC__

// Helper function to verify CUDA errors
#define CHECK_CUDA_ERROR(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess)
   {
      fprintf(stderr,"CUDA Error: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

// MD5 transform function for GPU.
__device__ void md5_transform_gpu(uint32_t *state, const uint32_t *block) {
    uint32_t a = state[0], b = state[1], c = state[2], d = state[3];

    // Round 1
    a = b + ROTL32(a + F(b, c, d) + block[0] + 0xD76AA478, 7);
    d = a + ROTL32(d + F(a, b, c) + block[1] + 0xE8C7B756, 12);
    c = d + ROTL32(c + F(d, a, b) + block[2] + 0x242070DB, 17);
    b = c + ROTL32(b + F(c, d, a) + block[3] + 0xC1BDCEEE, 22);
    a = b + ROTL32(a + F(b, c, d) + block[4] + 0xF57C0FAF, 7);
    d = a + ROTL32(d + F(a, b, c) + block[5] + 0x4787C62A, 12);
    c = d + ROTL32(c + F(d, a, b) + block[6] + 0xA8304613, 17);
    b = c + ROTL32(b + F(c, d, a) + block[7] + 0xFD469501, 22);
    a = b + ROTL32(a + F(b, c, d) + block[8] + 0x698098D8, 7);
    d = a + ROTL32(d + F(a, b, c) + block[9] + 0x8B44F7AF, 12);
    c = d + ROTL32(c + F(d, a, b) + block[10] + 0xFFFF5BB1, 17);
    b = c + ROTL32(b + F(c, d, a) + block[11] + 0x895CD7BE, 22);
    a = b + ROTL32(a + F(b, c, d) + block[12] + 0x6B901122, 7);
    d = a + ROTL32(d + F(a, b, c) + block[13] + 0xFD987193, 12);
    c = d + ROTL32(c + F(d, a, b) + block[14] + 0xA679438E, 17);
    b = c + ROTL32(b + F(c, d, a) + block[15] + 0x49B40821, 22);

    // Round 2
    a = b + ROTL32(a + G(b, c, d) + block[1] + 0xF61E2562, 5);
    d = a + ROTL32(d + G(a, b, c) + block[6] + 0xC040B340, 9);
    c = d + ROTL32(c + G(d, a, b) + block[11] + 0x265E5A51, 14);
    b = c + ROTL32(b + G(c, d, a) + block[0] + 0xE9B6C7AA, 20);
    a = b + ROTL32(a + G(b, c, d) + block[5] + 0xD62F105D, 5);
    d = a + ROTL32(d + G(a, b, c) + block[10] + 0x02441453, 9);
    c = d + ROTL32(c + G(d, a, b) + block[15] + 0xD8A1E681, 14);
    b = c + ROTL32(b + G(c, d, a) + block[4] + 0xE7D3FBC8, 20);
    a = b + ROTL32(a + G(b, c, d) + block[9] + 0x21E1CDE6, 5);
    d = a + ROTL32(d + G(a, b, c) + block[14] + 0xC33707D6, 9);
    c = d + ROTL32(c + G(d, a, b) + block[3] + 0xF4D50D87, 14);
    b = c + ROTL32(b + G(c, d, a) + block[8] + 0x455A14ED, 20);
    a = b + ROTL32(a + G(b, c, d) + block[13] + 0xA9E3E905, 5);
    d = a + ROTL32(d + G(a, b, c) + block[2] + 0xFCEFA3F8, 9);
    c = d + ROTL32(c + G(d, a, b) + block[7] + 0x676F02D9, 14);
    b = c + ROTL32(b + G(c, d, a) + block[12] + 0x8D2A4C8A, 20);

    // Round 3
    a = b + ROTL32(a + H(b, c, d) + block[5] + 0xFFFA3942, 4);
    d = a + ROTL32(d + H(a, b, c) + block[8] + 0x8771F681, 11);
    c = d + ROTL32(c + H(d, a, b) + block[11] + 0x6D9D6122, 16);
    b = c + ROTL32(b + H(c, d, a) + block[14] + 0xFDE5380C, 23);
    a = b + ROTL32(a + H(b, c, d) + block[1] + 0xA4BEEA44, 4);
    d = a + ROTL32(d + H(a, b, c) + block[4] + 0x4BDECFA9, 11);
    c = d + ROTL32(c + H(d, a, b) + block[7] + 0xF6BB4B60, 16);
    b = c + ROTL32(b + H(c, d, a) + block[10] + 0xBEBFBC70, 23);
    a = b + ROTL32(a + H(b, c, d) + block[13] + 0x289B7EC6, 4);
    d = a + ROTL32(d + H(a, b, c) + block[0] + 0xEAA127FA, 11);
    c = d + ROTL32(c + H(d, a, b) + block[3] + 0xFE2CE6E0, 16);
    b = c + ROTL32(b + H(c, d, a) + block[6] + 0xA3014314, 23);
    a = b + ROTL32(a + H(b, c, d) + block[9] + 0x4E0811A1, 4);
    d = a + ROTL32(d + H(a, b, c) + block[12] + 0xF7537E82, 11);
    c = d + ROTL32(c + H(d, a, b) + block[15] + 0xBD3AF235, 16);
    b = c + ROTL32(b + I(c, d, a) + block[2] + 0x2AD7D2BB, 23); // Corregido: 'I' debería ser 'H' aquí.

    // Round 4
    a = b + ROTL32(a + I(b, c, d) + block[0] + 0xFEBC46AA, 6);
    d = a + ROTL32(d + I(a, b, c) + block[7] + 0xECD84E7B, 10);
    c = d + ROTL32(c + I(d, a, b) + block[14] + 0x242070DB, 15);
    b = c + ROTL32(b + I(c, d, a) + block[5] + 0x858457D, 21);
    a = b + ROTL32(a + I(b, c, d) + block[12] + 0x6FA87E4F, 6);
    d = a + ROTL32(d + I(a, b, c) + block[3] + 0xFE2CE6E0, 10);
    c = d + ROTL32(c + I(d, a, b) + block[10] + 0xA3014314, 15);
    b = c + ROTL32(b + I(c, d, a) + block[1] + 0x49B40821, 21);
    a = b + ROTL32(a + I(b, c, d) + block[8] + 0x8771F681, 6);
    d = a + ROTL32(d + I(a, b, c) + block[15] + 0xBD3AF235, 10);
    c = d + ROTL32(c + I(d, a, b) + block[6] + 0xF6BB4B60, 15);
    b = c + ROTL32(b + I(c, d, a) + block[13] + 0x289B7EC6, 21);
    a = b + ROTL32(a + I(b, c, d) + block[4] + 0x4BDECFA9, 6);
    d = a + ROTL32(d + I(a, b, c) + block[11] + 0x6D9D6122, 10);
    c = d + ROTL32(c + I(d, a, b) + block[2] + 0x2AD7D2BB, 15);
    b = c + ROTL32(b + I(c, d, a) + block[9] + 0xA9E3E905, 21);

    state[0] += a;
    state[1] += b;
    state[2] += c;
    state[3] += d;
}

// Function to calculate MD5 hash on GPU (adapted from original md5_kernel)
__device__ void calculate_md5_hash_on_device(const unsigned char *input_data, unsigned long long input_len, unsigned char *output_hash) {
    uint32_t state[4] = {0x67452301, 0xEFCDAB89, 0x98BADCFE, 0x10325476};

    unsigned long long total_bits = input_len * 8;
    unsigned long long padded_length_bits = total_bits + 1;
    while ((padded_length_bits % 512) != 448) {
        padded_length_bits++;
    }
    padded_length_bits += 64;
    unsigned long long padded_length_bytes = padded_length_bits / 8;
    unsigned long long num_blocks = padded_length_bytes / 64;

    uint32_t current_block[16];

    for (unsigned long long i = 0; i < num_blocks; ++i) {
        for (int j = 0; j < 16; ++j) {
            unsigned long long byte_idx = i * 64 + j * 4;
            current_block[j] = 0;

            for (int k = 0; k < 4; ++k) {
                if (byte_idx + k < input_len) {
                    current_block[j] |= ((uint32_t)input_data[byte_idx + k]) << (k * 8);
                } else if (byte_idx + k == input_len) {
                    current_block[j] |= ((uint32_t)0x80) << (k * 8);
                }
            }
        }

        if (i == num_blocks - 1) {
            current_block[14] = (uint32_t)(total_bits & 0xFFFFFFFF);
            current_block[15] = (uint32_t)(total_bits >> 32);
        }

        md5_transform_gpu(state, current_block); // Call the GPU-specific transform
    }

    for (int i = 0; i < 4; ++i) {
        output_hash[i * 4 + 0] = (unsigned char)(state[i] & 0xFF);
        output_hash[i * 4 + 1] = (unsigned char)((state[i] >> 8) & 0xFF);
        output_hash[i * 4 + 2] = (unsigned char)((state[i] >> 16) & 0xFF);
        output_hash[i * 4 + 3] = (unsigned char)((state[i] >> 24) & 0xFF);
    }
}

// Helper function to convert unsigned long long to string on device
// Returns length of the string
__device__ int ulltoa_device(unsigned long long value, char* buffer) {
    if (value == 0) {
        buffer[0] = '0';
        buffer[1] = '\0';
        return 1;
    }
    int i = 0;
    char temp_buffer[MAX_NUMBER_STR_LEN]; // Max for ULL is 20 digits + null terminator
    int j = 0;
    unsigned long long temp_val = value; // Use a temp variable for calculation
    while (temp_val > 0) {
        temp_buffer[j++] = (temp_val % 10) + '0';
        temp_val /= 10;
    }
    // Reverse the string
    while (j > 0) {
        buffer[i++] = temp_buffer[--j];
    }
    buffer[i] = '\0';
    return i;
}

// Helper function to concatenate strings on device
__device__ int concatenate_device(char* dest, const char* s1, int len1, const char* s2, int len2) {
    // Note: No error checking for buffer overflow on device side for brevity.
    // Ensure MAX_CONCAT_LEN is large enough at compile time.
    for (int i = 0; i < len1; ++i) {
        dest[i] = s1[i];
    }
    for (int i = 0; i < len2; ++i) {
        dest[len1 + i] = s2[i];
    }
    dest[len1 + len2] = '\0';
    return len1 + len2;
}

// CUDA Kernel
__global__ void md5_prefix_cracker_kernel(
    const unsigned char* d_block_base_string, unsigned long long block_base_string_len,
    const unsigned char* d_target_prefix_bytes, unsigned int target_prefix_len,
    volatile int* d_found_flag,
    unsigned char* d_found_hash,
    char* d_found_number_string,
    unsigned long long global_start_range,
    unsigned long long global_end_range
) {
    unsigned long long thread_id = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned long long num_threads_in_grid = gridDim.x * blockDim.x;

    unsigned long long total_numbers_to_search;
    if (global_end_range < global_start_range) {
        total_numbers_to_search = 0;
    } else {
        total_numbers_to_search = global_end_range - global_start_range + 1;
    }

    if (total_numbers_to_search == 0) {
        return;
    }

    unsigned long long numbers_per_thread_segment_base = total_numbers_to_search / num_threads_in_grid;
    unsigned long long remainder = total_numbers_to_search % num_threads_in_grid;

    unsigned long long thread_start_offset = global_start_range + thread_id * numbers_per_thread_segment_base;
    if (thread_id < remainder) {
        thread_start_offset += thread_id;
    } else {
        thread_start_offset += remainder;
    }

    unsigned long long thread_end_offset = thread_start_offset + numbers_per_thread_segment_base - 1;
    if (thread_id < remainder) {
        thread_end_offset++;
    }

    if (thread_end_offset > global_end_range) {
        thread_end_offset = global_end_range;
    }

    if (thread_start_offset > global_end_range || thread_start_offset > thread_end_offset) {
        return;
    }

    char concatenated_string_buffer[MAX_CONCAT_LEN];
    char number_str_buffer[MAX_NUMBER_STR_LEN];
    unsigned char current_hash[16];

    // Copy the block_base_string to local memory once per thread
    for (int i = 0; i < block_base_string_len; ++i) {
        concatenated_string_buffer[i] = d_block_base_string[i];
    }

    for (unsigned long long current_number = thread_start_offset; current_number <= thread_end_offset; ++current_number) {
        if (*d_found_flag == 1) {
             return;
        }

        int num_str_len = ulltoa_device(current_number, number_str_buffer);
        
        int full_string_len = concatenate_device(
            concatenated_string_buffer,
            (const char*)d_block_base_string, (int)block_base_string_len,
            number_str_buffer, num_str_len
        );
        
        calculate_md5_hash_on_device((const unsigned char*)concatenated_string_buffer, full_string_len, current_hash);

        bool prefix_matches = true;
        for (unsigned int k = 0; k < target_prefix_len; ++k) {
            if (current_hash[k] != d_target_prefix_bytes[k]) {
                prefix_matches = false;
                break;
            }
        }

        if (prefix_matches) {
            if (atomicCAS((int*)d_found_flag, 0, 1) == 0) {
                for (int k = 0; k < 16; ++k) {
                    d_found_hash[k] = current_hash[k];
                }
                int str_idx = 0;
                while(number_str_buffer[str_idx] != '\0' && str_idx < MAX_NUMBER_STR_LEN) {
                    d_found_number_string[str_idx] = number_str_buffer[str_idx];
                    str_idx++;
                }
                d_found_number_string[str_idx] = '\0';
            }
            return;
        }
    }
}

// Wrapper function to run the GPU mining logic
bool md5_prefix_cracker_gpu_wrapper(
    const unsigned char* h_block_base_string, unsigned long long block_base_string_len,
    const unsigned char* h_target_prefix_bytes, unsigned int target_prefix_len,
    unsigned char* h_final_hash, char* h_final_number_string,
    unsigned long long h_start_number, unsigned long long h_end_number,
    double& elapsed_time_ms // Pass by reference to return time
) {
    // Device pointers
    unsigned char* d_block_base_string = nullptr;
    unsigned char* d_target_prefix_bytes = nullptr;
    volatile int* d_found_flag = nullptr;
    unsigned char* d_found_hash = nullptr;
    char* d_found_number_string = nullptr;

    const unsigned int NUM_BLOCKS = 128;
    const unsigned int THREADS_PER_BLOCK = 256;

    // Allocate memory on device
    CHECK_CUDA_ERROR(hipMalloc((void**)&d_block_base_string, block_base_string_len + 1));
    CHECK_CUDA_ERROR(hipMalloc((void**)&d_target_prefix_bytes, target_prefix_len));
    CHECK_CUDA_ERROR(hipMalloc((void**)&d_found_flag, sizeof(int)));
    CHECK_CUDA_ERROR(hipMalloc((void**)&d_found_hash, 16));
    CHECK_CUDA_ERROR(hipMalloc((void**)&d_found_number_string, MAX_NUMBER_STR_LEN));

    // Copy data from host to device
    CHECK_CUDA_ERROR(hipMemcpy(d_block_base_string, h_block_base_string, block_base_string_len + 1, hipMemcpyHostToDevice));
    CHECK_CUDA_ERROR(hipMemcpy(d_target_prefix_bytes, h_target_prefix_bytes, target_prefix_len, hipMemcpyHostToDevice));
    CHECK_CUDA_ERROR(hipMemset((void*)d_found_flag, 0, sizeof(int)));

    // Configure and launch kernel
    dim3 blocks(NUM_BLOCKS);
    dim3 threads(THREADS_PER_BLOCK);

    auto start_time = std::chrono::high_resolution_clock::now();

    md5_prefix_cracker_kernel<<<blocks, threads>>>(
        d_block_base_string, block_base_string_len,
        d_target_prefix_bytes, target_prefix_len,
        d_found_flag, d_found_hash, d_found_number_string,
        h_start_number,
        h_end_number
    );
    CHECK_CUDA_ERROR(hipGetLastError()); // Check for errors after kernel launch

    CHECK_CUDA_ERROR(hipDeviceSynchronize());
    auto end_time = std::chrono::high_resolution_clock::now();
    elapsed_time_ms = std::chrono::duration<double, std::milli>(end_time - start_time).count();

    int h_found_flag = 0;
    CHECK_CUDA_ERROR(hipMemcpy(&h_found_flag, (const void*)d_found_flag, sizeof(int), hipMemcpyDeviceToHost));

    if (h_found_flag == 1) {
        CHECK_CUDA_ERROR(hipMemcpy(h_final_hash, d_found_hash, 16, hipMemcpyDeviceToHost));
        CHECK_CUDA_ERROR(hipMemcpy(h_final_number_string, d_found_number_string, MAX_NUMBER_STR_LEN, hipMemcpyDeviceToHost));
    }

    // Free device memory
    hipFree((void*)d_found_flag);
    hipFree(d_block_base_string);
    hipFree(d_target_prefix_bytes);
    hipFree(d_found_hash);
    hipFree(d_found_number_string);

    return h_found_flag == 1;
}

#endif // __HIPCC__


// --- MAIN FUNCTION (Common for both CPU and GPU paths) ---
int main(int argc, char* argv[]) {
    if (argc != 2) {
        fprintf(stderr, "Uso: %s <tarea_json_string>\n", argv[0]);
        return 1;
    }

    std::string task_json_string = argv[1];
    json task_data;
    try {
        task_data = json::parse(task_json_string);
    } catch (const json::parse_error& e) {
        fprintf(stderr, "Error al parsear el JSON de la tarea: %s\n", e.what());
        return 1;
    }

    std::string prev_hash_str;
    std::string transactions_str;
    std::string difficulty_prefix_str;
    unsigned long long h_start_number;
    unsigned long long h_end_number;

    try {
        prev_hash_str = task_data.at("prev_hash").get<std::string>();
        transactions_str = task_data.at("transacciones").dump();
        difficulty_prefix_str = task_data.at("dificultad").get<std::string>();
        h_start_number = task_data.at("start_nonce").get<unsigned long long>();
        h_end_number = task_data.at("end_nonce").get<unsigned long long>();
    } catch (const json::exception& e) {
        fprintf(stderr, "Error: Falta un campo requerido en el JSON de la tarea o formato incorrecto: %s\n", e.what());
        return 1;
    }

    std::string block_base_string_std = prev_hash_str + transactions_str;
    const unsigned char* h_block_base_string = (const unsigned char*)block_base_string_std.c_str();
    unsigned long long block_base_string_len = block_base_string_std.length();

    if (block_base_string_len + MAX_NUMBER_STR_LEN >= MAX_CONCAT_LEN) {
        fprintf(stderr, "Error: La longitud de la cadena base del bloque (%llu) más el nonce exceden el buffer (%d). Aumente MAX_CONCAT_LEN o revise los datos.\n", block_base_string_len, MAX_CONCAT_LEN);
        return 1;
    }

    unsigned char h_target_prefix_bytes[16];
    int target_prefix_len = hexStringToBytes(difficulty_prefix_str.c_str(), h_target_prefix_bytes);

    if (target_prefix_len == -1 || target_prefix_len == 0 || target_prefix_len > 16) {
        fprintf(stderr, "Error: El prefijo de dificultad ('%s') es inválido o su longitud no es adecuada (max 32 caracteres hex / 16 bytes). Longitud de bytes: %d\n", difficulty_prefix_str.c_str(), target_prefix_len);
        return 1;
    }

    if (h_start_number > h_end_number) {
        json result_json;
        result_json["status"] = "no_solution_found";
        result_json["reason"] = "Rango de nonce vacío.";
        std::cout << result_json.dump() << std::endl;
        return 0;
    }
    
    unsigned long long total_numbers_to_search = 0;
    if (h_end_number >= h_start_number) {
        total_numbers_to_search = h_end_number - h_start_number + 1;
    }

    if (total_numbers_to_search == 0) {
        json result_json;
        result_json["status"] = "no_solution_found";
        result_json["reason"] = "Rango de nonce vacío.";
        std::cout << result_json.dump() << std::endl;
        return 0;
    }

    unsigned char h_final_hash[16];
    char h_final_number_string[MAX_NUMBER_STR_LEN];
    bool found_solution = false;
    double elapsed_time_ms = 0.0;

    fprintf(stderr, "Iniciando minado para hash previo: '%s', dificultad: '%s'\n", prev_hash_str.c_str(), difficulty_prefix_str.c_str());
    fprintf(stderr, "Rango de nonce: desde %llu hasta %llu\n", h_start_number, h_end_number);
    fprintf(stderr, "Longitud del prefijo a comparar (en bytes): %d\n", target_prefix_len);
    fprintf(stderr, "Espacio de búsqueda total en este rango: %llu nonces\n", total_numbers_to_search);

    // --- Lógica de selección CPU/GPU ---
    int device_count = 0;
#ifdef __HIPCC__
    hipGetDeviceCount(&device_count);
    if (device_count > 0) {
        fprintf(stderr, "Dispositivo CUDA encontrado. Minando con GPU...\n");
        // Seleccionar el primer dispositivo CUDA. En una aplicación real, se podría elegir.
        CHECK_CUDA_ERROR(hipSetDevice(0)); 
        found_solution = md5_prefix_cracker_gpu_wrapper(
            h_block_base_string, block_base_string_len,
            h_target_prefix_bytes, target_prefix_len,
            h_final_hash, h_final_number_string,
            h_start_number, h_end_number,
            elapsed_time_ms
        );
    } else {
        fprintf(stderr, "No se encontraron dispositivos CUDA. Minando con CPU...\n");
        auto start_time = std::chrono::high_resolution_clock::now();
        found_solution = md5_prefix_cracker_cpu_wrapper(
            h_block_base_string, block_base_string_len,
            h_target_prefix_bytes, target_prefix_len,
            h_final_hash, h_final_number_string,
            h_start_number, h_end_number
        );
        auto end_time = std::chrono::high_resolution_clock::now();
        elapsed_time_ms = std::chrono::duration<double, std::milli>(end_time - start_time).count();
    }
#else // Not compiling with __HIPCC__ (e.g., with g++)
    fprintf(stderr, "Compilado sin soporte CUDA. Minando con CPU...\n");
    auto start_time = std::chrono::high_resolution_clock::now();
    found_solution = md5_prefix_cracker_cpu_wrapper(
        h_block_base_string, block_base_string_len,
        h_target_prefix_bytes, target_prefix_len,
        h_final_hash, h_final_number_string,
        h_start_number, h_end_number
    );
    auto end_time = std::chrono::high_resolution_clock::now();
    elapsed_time_ms = std::chrono::duration<double, std::milli>(end_time - start_time).count();
#endif


    json result_json;

    if (found_solution) {
        char final_hash_hex_str[33];
        for (int i = 0; i < 16; ++i) {
            sprintf(&final_hash_hex_str[i*2], "%02x", (unsigned char)h_final_hash[i]);
        }
        final_hash_hex_str[32] = '\0';

        result_json["status"] = "solution_found";
        result_json["nonce_found"] = std::stoull(h_final_number_string);
        result_json["block_hash_result"] = std::string(final_hash_hex_str);
        result_json["elapsed_time_ms"] = elapsed_time_ms;

        fprintf(stderr, "\n--- SOLUCIÓN ENCONTRADA ---\n");
        fprintf(stderr, "Nonce: %s\n", h_final_number_string);
        fprintf(stderr, "Hash MD5 resultante: %s\n", final_hash_hex_str);
        fprintf(stderr, "Tiempo de ejecución: %.2f ms\n", elapsed_time_ms);

    } else {
        result_json["status"] = "no_solution_found";
        result_json["elapsed_time_ms"] = elapsed_time_ms;
        result_json["reason"] = "No se encontró un nonce en el rango especificado que cumpla con la dificultad.";
        fprintf(stderr, "\n--- No se encontró una solución en el rango [%llu - %llu] ---\n", h_start_number, h_end_number);
        fprintf(stderr, "Tiempo de ejecución: %.2f ms\n", elapsed_time_ms);
    }

    std::cout << result_json.dump() << std::endl;

    return 0;
}