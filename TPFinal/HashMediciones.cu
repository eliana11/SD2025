#include <hip/hip_runtime.h>
#include <stdio.h>
#include <string.h>
#include <iostream>
#include <stdint.h>
#include <assert.h>
#include <limits.h>
#include <chrono> // <-- Asegúrate de tener este include

// ***********************************************************************************
// INICIO DEL CÓDIGO DE LA LIBRERÍA MD5 EN CUDA (md5.cu de honours-project)
// (CORREGIDO DE ERRORES DE SINTAXIS)
// ***********************************************************************************

// MD5 basic functions.
#define F(x, y, z) (((x) & (y)) | ((~x) & (z)))
#define G(x, y, z) (((x) & (z)) | ((y) & (~z)))
#define H(x, y, z) ((x) ^ (y) ^ (z))
#define I(x, y, z) ((y) ^ ((x) | (~z)))

// Rotate Left function.
#define ROTL32(x, n) (((x) << (n)) | ((x) >> (32 - (n))))

// MD5 transform function.
// All values are uint32_t.
__device__ void md5_transform(uint32_t *state, const uint32_t *block) {
    uint32_t a = state[0], b = state[1], c = state[2], d = state[3];

    // Round 1
    a = b + ROTL32(a + F(b, c, d) + block[0] + 0xD76AA478, 7);
    d = a + ROTL32(d + F(a, b, c) + block[1] + 0xE8C7B756, 12);
    c = d + ROTL32(c + F(d, a, b) + block[2] + 0x242070DB, 17);
    b = c + ROTL32(b + F(c, d, a) + block[3] + 0xC1BDCEEE, 22);
    a = b + ROTL32(a + F(b, c, d) + block[4] + 0xF57C0FAF, 7);
    d = a + ROTL32(d + F(a, b, c) + block[5] + 0x4787C62A, 12);
    c = d + ROTL32(c + F(d, a, b) + block[6] + 0xA8304613, 17);
    b = c + ROTL32(b + F(c, d, a) + block[7] + 0xFD469501, 22);
    a = b + ROTL32(a + F(b, c, d) + block[8] + 0x698098D8, 7);
    d = a + ROTL32(d + F(a, b, c) + block[9] + 0x8B44F7AF, 12);
    c = d + ROTL32(c + F(d, a, b) + block[10] + 0xFFFF5BB1, 17);
    b = c + ROTL32(b + F(c, d, a) + block[11] + 0x895CD7BE, 22);
    a = b + ROTL32(a + F(b, c, d) + block[12] + 0x6B901122, 7);
    d = a + ROTL32(d + F(a, b, c) + block[13] + 0xFD987193, 12);
    c = d + ROTL32(c + F(d, a, b) + block[14] + 0xA679438E, 17);
    b = c + ROTL32(b + F(c, d, a) + block[15] + 0x49B40821, 22);

    // Round 2
    a = b + ROTL32(a + G(b, c, d) + block[1] + 0xF61E2562, 5);
    d = a + ROTL32(d + G(a, b, c) + block[6] + 0xC040B340, 9);
    c = d + ROTL32(c + G(d, a, b) + block[11] + 0x265E5A51, 14);
    b = c + ROTL32(b + G(c, d, a) + block[0] + 0xE9B6C7AA, 20);
    a = b + ROTL32(a + G(b, c, d) + block[5] + 0xD62F105D, 5);
    d = a + ROTL32(d + G(a, b, c) + block[10] + 0x02441453, 9);
    c = d + ROTL32(c + G(d, a, b) + block[15] + 0xD8A1E681, 14);
    b = c + ROTL32(b + G(c, d, a) + block[4] + 0xE7D3FBC8, 20);
    a = b + ROTL32(a + G(b, c, d) + block[9] + 0x21E1CDE6, 5);
    d = a + ROTL32(d + G(a, b, c) + block[14] + 0xC33707D6, 9);
    c = d + ROTL32(c + G(d, a, b) + block[3] + 0xF4D50D87, 14);
    b = c + ROTL32(b + G(c, d, a) + block[8] + 0x455A14ED, 20);
    a = b + ROTL32(a + G(b, c, d) + block[13] + 0xA9E3E905, 5);
    d = a + ROTL32(d + G(a, b, c) + block[2] + 0xFCEFA3F8, 9);
    c = d + ROTL32(c + G(d, a, b) + block[7] + 0x676F02D9, 14);
    b = c + ROTL32(b + G(c, d, a) + block[12] + 0x8D2A4C8A, 20);

    // Round 3
    a = b + ROTL32(a + H(b, c, d) + block[5] + 0xFFFA3942, 4);
    d = a + ROTL32(d + H(a, b, c) + block[8] + 0x8771F681, 11);
    c = d + ROTL32(c + H(d, a, b) + block[11] + 0x6D9D6122, 16);
    b = c + ROTL32(b + H(c, d, a) + block[14] + 0xFDE5380C, 23);
    a = b + ROTL32(a + H(b, c, d) + block[1] + 0xA4BEEA44, 4);
    d = a + ROTL32(d + H(a, b, c) + block[4] + 0x4BDECFA9, 11);
    c = d + ROTL32(c + H(d, a, b) + block[7] + 0xF6BB4B60, 16);
    b = c + ROTL32(b + H(c, d, a) + block[10] + 0xBEBFBC70, 23);
    a = b + ROTL32(a + H(b, c, d) + block[13] + 0x289B7EC6, 4);
    d = a + ROTL32(d + H(a, b, c) + block[0] + 0xEAA127FA, 11);
    c = d + ROTL32(c + H(d, a, b) + block[3] + 0xFE2CE6E0, 16);
    b = c + ROTL32(b + H(c, d, a) + block[6] + 0xA3014314, 23);
    a = b + ROTL32(a + H(b, c, d) + block[9] + 0x4E0811A1, 4);
    d = a + ROTL32(d + H(a, b, c) + block[12] + 0xF7537E82, 11);
    c = d + ROTL32(c + H(d, a, b) + block[15] + 0xBD3AF235, 16);
    b = c + ROTL32(b + H(c, d, a) + block[2] + 0x2AD7D2BB, 23);

    // Round 4
    a = b + ROTL32(a + I(b, c, d) + block[0] + 0xFEBC46AA, 6);
    d = a + ROTL32(d + I(a, b, c) + block[7] + 0xECD84E7B, 10);
    c = d + ROTL32(c + I(d, a, b) + block[14] + 0x242070DB, 15);
    b = c + ROTL32(b + I(c, d, a) + block[5] + 0x858457D, 21);
    a = b + ROTL32(a + I(b, c, d) + block[12] + 0x6FA87E4F, 6);
    d = a + ROTL32(d + I(a, b, c) + block[3] + 0xFE2CE6E0, 10);
    c = d + ROTL32(c + I(d, a, b) + block[10] + 0xA3014314, 15);
    b = c + ROTL32(b + I(c, d, a) + block[1] + 0x49B40821, 21);
    a = b + ROTL32(a + I(b, c, d) + block[8] + 0x8771F681, 6);
    d = a + ROTL32(d + I(a, b, c) + block[15] + 0xBD3AF235, 10);
    c = d + ROTL32(c + I(d, a, b) + block[6] + 0xF6BB4B60, 15);
    b = c + ROTL32(b + I(c, d, a) + block[13] + 0x289B7EC6, 21);
    a = b + ROTL32(a + I(b, c, d) + block[4] + 0x4BDECFA9, 6);
    d = a + ROTL32(d + I(a, b, c) + block[11] + 0x6D9D6122, 10);
    c = d + ROTL32(c + I(d, a, b) + block[2] + 0x2AD7D2BB, 15);
    b = c + ROTL32(b + I(c, d, a) + block[9] + 0xA9E3E905, 21);

    state[0] += a;
    state[1] += b;
    state[2] += c;
    state[3] += d;
}

// Function to calculate MD5 hash on GPU (adapted from original md5_kernel)
// This will be called from within the main cracking kernel.
__device__ void calculate_md5_hash_on_device(const unsigned char *input_data, unsigned long long input_len, unsigned char *output_hash) {
    uint32_t state[4] = {0x67452301, 0xEFCDAB89, 0x98BADCFE, 0x10325476};

    unsigned long long total_bits = input_len * 8;
    unsigned long long padded_length_bits = total_bits + 1;
    while ((padded_length_bits % 512) != 448) {
        padded_length_bits++;
    }
    padded_length_bits += 64;
    unsigned long long padded_length_bytes = padded_length_bits / 8;
    unsigned long long num_blocks = padded_length_bytes / 64;

    uint32_t current_block[16];

    for (unsigned long long i = 0; i < num_blocks; ++i) {
        for (int j = 0; j < 16; ++j) {
            unsigned long long byte_idx = i * 64 + j * 4;
            current_block[j] = 0;

            for (int k = 0; k < 4; ++k) {
                if (byte_idx + k < input_len) {
                    current_block[j] |= ((uint32_t)input_data[byte_idx + k]) << (k * 8);
                } else if (byte_idx + k == input_len) {
                    current_block[j] |= ((uint32_t)0x80) << (k * 8);
                }
            }
        }

        if (i == num_blocks - 1) {
            current_block[14] = (uint32_t)(total_bits & 0xFFFFFFFF);
            current_block[15] = (uint32_t)(total_bits >> 32);
        }

        md5_transform(state, current_block);
    }

    for (int i = 0; i < 4; ++i) {
        output_hash[i * 4 + 0] = (unsigned char)(state[i] & 0xFF);
        output_hash[i * 4 + 1] = (unsigned char)((state[i] >> 8) & 0xFF);
        output_hash[i * 4 + 2] = (unsigned char)((state[i] >> 16) & 0xFF);
        output_hash[i * 4 + 3] = (unsigned char)((state[i] >> 24) & 0xFF);
    }
}

// ***********************************************************************************
// FIN DEL CÓDIGO DE LA LIBRERÍA MD5 EN CUDA
// ***********************************************************************************

// --- NUEVAS FUNCIONES Y KERNEL PARA LA BÚSQUEDA ---

// Helper function to convert unsigned long long to string on device
// Returns length of the string
__device__ int ulltoa_device(unsigned long long value, char* buffer) {
    if (value == 0) {
        buffer[0] = '0';
        buffer[1] = '\0';
        return 1;
    }
    int i = 0;
    char temp_buffer[20]; // Max for ULL is 20 digits + null terminator
    int j = 0;
    unsigned long long temp_val = value; // Use a temp variable for calculation
    while (temp_val > 0) {
        temp_buffer[j++] = (temp_val % 10) + '0';
        temp_val /= 10;
    }
    // Reverse the string
    while (j > 0) {
        buffer[i++] = temp_buffer[--j];
    }
    buffer[i] = '\0';
    return i;
}

// Helper function to concatenate strings on device
__device__ int concatenate_device(char* dest, const char* s1, int len1, const char* s2, int len2) {
    for (int i = 0; i < len1; ++i) {
        dest[i] = s1[i];
    }
    for (int i = 0; i < len2; ++i) {
        dest[len1 + i] = s2[i];
    }
    dest[len1 + len2] = '\0';
    return len1 + len2;
}

// Maximum length for the concatenated string (base_string + number_string)
// Assuming base_string max 200 chars, number max 10^18 (19 chars), plus null terminator
#define MAX_CONCAT_LEN 256
// Max length for the number string (e.g., 10^18 is 19 digits)
#define MAX_NUMBER_STR_LEN 20

// Kernel para la búsqueda del prefijo MD5
// d_base_string: cadena base en la GPU
// base_string_len: longitud de la cadena base
// d_target_prefix_bytes: prefijo MD5 objetivo en bytes
// target_prefix_len: longitud del prefijo objetivo en bytes
// d_found_flag: puntero a una bandera atómica (1 si se encontró, 0 en otro caso)
// d_found_hash: puntero a donde almacenar el hash encontrado
// d_found_number_string: puntero a donde almacenar el número encontrado como string
// start_num_offset: número inicial para la búsqueda del hilo (offset para el ID del hilo)
// max_attempts_per_thread: máximo de números que cada hilo intentará
__global__ void md5_prefix_cracker_kernel(
    const unsigned char* d_base_string, unsigned long long base_string_len,
    const unsigned char* d_target_prefix_bytes, unsigned int target_prefix_len,
    volatile int* d_found_flag,
    unsigned char* d_found_hash,
    char* d_found_number_string,
    unsigned long long start_num_offset,
    unsigned long long max_attempts_per_thread)
{
    unsigned long long thread_id = blockIdx.x * blockDim.x + threadIdx.x;

    // Local buffer for the concatenated string
    char concatenated_string_buffer[MAX_CONCAT_LEN];
    char number_str_buffer[MAX_NUMBER_STR_LEN];
    unsigned char current_hash[16]; // To store the hash calculated by this thread

    // Each thread starts its search from a unique number and iterates
    // This allows many threads to search in parallel
    for (unsigned long long i = 0; i < max_attempts_per_thread; ++i) {
        // Check if a solution has already been found by another thread.
        // A volatile read is sufficient here for early exit for performance.
        if (*d_found_flag == 1) {
             return; // Another thread found it, so this thread exits
        }

        unsigned long long current_number = start_num_offset + (thread_id * max_attempts_per_thread) + i;

        // Convert number to string
        int num_str_len = ulltoa_device(current_number, number_str_buffer);

        // Concatenate base_string and number_string
        int full_string_len = concatenate_device(
            concatenated_string_buffer,
            (const char*)d_base_string, (int)base_string_len, // Cast to int for concatenate_device
            number_str_buffer, num_str_len
        );

        // Calculate MD5 hash for the concatenated string
        calculate_md5_hash_on_device((const unsigned char*)concatenated_string_buffer, full_string_len, current_hash);

        // Compare the calculated hash prefix with the target prefix
        bool prefix_matches = true;
        for (unsigned int k = 0; k < target_prefix_len; ++k) {
            if (current_hash[k] != d_target_prefix_bytes[k]) {
                prefix_matches = false;
                break;
            }
        }

        // If prefix matches, atomically update found_flag and store results
        if (prefix_matches) {
            // Use atomicCAS to ensure only the first thread to find a match writes the result
            if (atomicCAS((int*)d_found_flag, 0, 1) == 0) { // If it was 0, set to 1 (this thread is the first)
                // Copy the found hash
                for (int k = 0; k < 16; ++k) {
                    d_found_hash[k] = current_hash[k];
                }
                // Copy the found number string
                int str_idx = 0;
                while(number_str_buffer[str_idx] != '\0' && str_idx < MAX_NUMBER_STR_LEN) {
                    d_found_number_string[str_idx] = number_str_buffer[str_idx];
                    str_idx++;
                }
                d_found_number_string[str_idx] = '\0'; // Null terminate
            }
            return; // Exit kernel for this thread after finding/confirming a match
        }
    }
}


// --- FUNCIONES DE AYUDA DEL HOST ---

// Función de ayuda para verificar errores de CUDA
#define CHECK_CUDA_ERROR(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess)
   {
      fprintf(stderr,"CUDA Error: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

// Converts a hexadecimal character to its integer value
int hexCharToInt(char c) {
    if (c >= '0' && c <= '9') return c - '0';
    if (c >= 'a' && c <= 'f') return c - 'a' + 10;
    if (c >= 'A' && c <= 'F') return c - 'A' + 10;
    return -1; // Invalid hex char
}

// Converts a hexadecimal string to a byte array
// Returns the number of bytes written to byte_array, or -1 on error
int hexStringToBytes(const char* hex_string, unsigned char* byte_array) {
    int len = strlen(hex_string);
    if (len % 2 != 0) {
        // Hex string must have an even number of characters
        // Or if it's a single hex digit, convert it to 0X style
        if (len == 1) { // Handle "1" becoming "01"
            int nibble = hexCharToInt(hex_string[0]);
            if (nibble == -1) return -1;
            byte_array[0] = (unsigned char)nibble; // Will be 0X
            return 1;
        }
        return -1;
    }
    int byte_len = len / 2;
    for (int i = 0; i < byte_len; ++i) {
        int high_nibble = hexCharToInt(hex_string[i * 2]);
        int low_nibble = hexCharToInt(hex_string[i * 2 + 1]);
        if (high_nibble == -1 || low_nibble == -1) {
            return -1; // Invalid hex character
        }
        byte_array[i] = (high_nibble << 4) | low_nibble;
    }
    return byte_len;
}


int main(int argc, char* argv[]) {
    if (argc != 3) {
        fprintf(stderr, "Uso: %s <hash_prefix_hex> <base_string>\n", argv[0]);
        return 1;
    }

    const char* h_hash_prefix_hex = argv[1];
    const unsigned char* h_base_string = (const unsigned char*)argv[2];

    unsigned char h_target_prefix_bytes[16]; // Max MD5 hash size
    int target_prefix_len = hexStringToBytes(h_hash_prefix_hex, h_target_prefix_bytes);

    if (target_prefix_len == -1 || target_prefix_len == 0 || target_prefix_len > 16) {
        fprintf(stderr, "Error: El prefijo del hash hexadecimal es inválido o su longitud no es adecuada (max 32 caracteres hex / 16 bytes). Recibido: %s, Longitud de bytes: %d\n", h_hash_prefix_hex, target_prefix_len);
        return 1;
    }
    
    unsigned long long base_string_len = strlen((const char*)h_base_string);
    if (base_string_len + MAX_NUMBER_STR_LEN >= MAX_CONCAT_LEN) {
        fprintf(stderr, "Error: La longitud de la cadena base (%llu) más el número exceden el buffer (%d). Aumente MAX_CONCAT_LEN o reduzca la cadena base.\n", base_string_len, MAX_CONCAT_LEN);
        return 1;
    }

    // --- Variables en el Host (punteros a memoria del Device) ---
    unsigned char* d_base_string = nullptr;
    unsigned char* d_target_prefix_bytes = nullptr;
    volatile int* d_found_flag = nullptr; // Bandera para indicar si se encontró el hash
    unsigned char* d_found_hash = nullptr; // Hash encontrado
    char* d_found_number_string = nullptr; // Número (como string) que generó el hash

    // --- Parámetros de Configuración del Kernel ---
    // Puedes ajustar estos valores para más o menos hilos/intentos.
    // Cuanto más largo el prefijo o más "raro", más intentos necesitará.
    const unsigned int NUM_BLOCKS = 128; // Número de bloques
    const unsigned int THREADS_PER_BLOCK = 256; // Hilos por bloque
    // Este valor determina cuántos números consecutivos probará CADA hilo.
    // Multiplicado por NUM_BLOCKS * THREADS_PER_BLOCK da el espacio total de búsqueda.
    const unsigned long long MAX_ATTEMPTS_PER_THREAD = 10000; // Máximo de números a probar por CADA hilo
    const unsigned long long TOTAL_SEARCH_SPACE = (unsigned long long)NUM_BLOCKS * THREADS_PER_BLOCK * MAX_ATTEMPTS_PER_THREAD; // <-- CORREGIDO NUM_BLOCKs a NUM_BLOCKS

    printf("Iniciando búsqueda de prefijo MD5: '%s' para la cadena base '%s'\n", h_hash_prefix_hex, h_base_string);
    printf("Longitud del prefijo a comparar (en bytes): %d\n", target_prefix_len);
    printf("Espacio de búsqueda total (aproximado): %llu números\n", TOTAL_SEARCH_SPACE);
    printf("--- Por favor, espere, esto puede tardar ---\n");


    // 1. Asignar memoria en el device
    CHECK_CUDA_ERROR(hipMalloc((void**)&d_base_string, base_string_len + 1));
    CHECK_CUDA_ERROR(hipMalloc((void**)&d_target_prefix_bytes, target_prefix_len));
    CHECK_CUDA_ERROR(hipMalloc((void**)&d_found_flag, sizeof(int)));
    CHECK_CUDA_ERROR(hipMalloc((void**)&d_found_hash, 16)); // MD5 hash es de 16 bytes
    CHECK_CUDA_ERROR(hipMalloc((void**)&d_found_number_string, MAX_NUMBER_STR_LEN));

    // 2. Copiar datos del host al device
    CHECK_CUDA_ERROR(hipMemcpy(d_base_string, h_base_string, base_string_len + 1, hipMemcpyHostToDevice));
    CHECK_CUDA_ERROR(hipMemcpy(d_target_prefix_bytes, h_target_prefix_bytes, target_prefix_len, hipMemcpyHostToDevice));
    CHECK_CUDA_ERROR(hipMemset((void*)d_found_flag, 0, sizeof(int)));

    // 3. Configurar y lanzar el kernel de búsqueda
    dim3 blocks(NUM_BLOCKS);
    dim3 threads(THREADS_PER_BLOCK);

    // El número inicial para el primer hilo (0) es 0. Cada hilo calcula su propio offset.
    unsigned long long start_num_for_kernel = 0;

    // --- INICIO MEDICIÓN DE TIEMPO ---
    auto start_time = std::chrono::high_resolution_clock::now();

    md5_prefix_cracker_kernel<<<blocks, threads>>>(
        d_base_string, base_string_len,
        d_target_prefix_bytes, target_prefix_len,
        d_found_flag, d_found_hash, d_found_number_string,
        start_num_for_kernel, MAX_ATTEMPTS_PER_THREAD
    );
    CHECK_CUDA_ERROR(hipGetLastError()); // Verifica si hubo error en el lanzamiento del kernel

    // 4. Sincronizar el dispositivo y finalizar la medición
    CHECK_CUDA_ERROR(hipDeviceSynchronize()); // Espera a que todos los hilos terminen
    auto end_time = std::chrono::high_resolution_clock::now();
    // --- FIN MEDICIÓN DE TIEMPO ---

    std::chrono::duration<double, std::milli> duration = end_time - start_time; // Duración en milisegundos

    // 5. Verificar si se encontró una solución y copiar resultados
    int h_found_flag = 0;
    CHECK_CUDA_ERROR(hipMemcpy(&h_found_flag, (const void*)d_found_flag, sizeof(int), hipMemcpyDeviceToHost));

    if (h_found_flag == 1) {
        unsigned char h_final_hash[16];
        char h_final_number_string[MAX_NUMBER_STR_LEN];
        CHECK_CUDA_ERROR(hipMemcpy(h_final_hash, d_found_hash, 16, hipMemcpyDeviceToHost));
        CHECK_CUDA_ERROR(hipMemcpy(h_final_number_string, d_found_number_string, MAX_NUMBER_STR_LEN, hipMemcpyDeviceToHost));

        printf("\n--- SOLUCIÓN ENCONTRADA ---\n");
        printf("Número encontrado: %s\n", h_final_number_string);
        printf("Cadena probada: %s%s\n", (const char*)h_base_string, h_final_number_string);
        printf("Hash MD5 resultante: ");
        for (int i = 0; i < 16; ++i) {
            printf("%02x", (unsigned char)h_final_hash[i]);
        }
        printf("\n");
        printf("Tiempo de ejecución: %.2f ms\n", duration.count()); // Imprimir el tiempo
    } else {
        printf("\n--- No se encontró una solución en el rango de búsqueda especificado (%llu intentos). ---\n", TOTAL_SEARCH_SPACE);
        printf("Intente aumentar MAX_ATTEMPTS_PER_THREAD o NUM_BLOCKS.\n");
        printf("Tiempo de ejecución: %.2f ms\n", duration.count()); // Imprimir el tiempo incluso si no se encontró
    }

    // 6. Liberar memoria
    hipFree((void*)d_found_flag);
    hipFree(d_base_string);
    hipFree(d_target_prefix_bytes);
    hipFree(d_found_hash);
    hipFree(d_found_number_string);

    return 0;
}
