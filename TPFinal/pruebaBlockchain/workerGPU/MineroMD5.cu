#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <sstream>
#include <string>
#include <chrono>
#include <regex> // Para el regex en el host (CPU)
#include <limits.h> // Para ULLONG_MAX
#include <cstring>  // Para memcpy en el host, aunque no se usa directamente ahora
#include <cstdio>   // Para sprintf
#include <algorithm> // Para std::min

// Incluisiones de CUDA
#include <hip/hip_runtime.h>
#include <>

// Incluye la librería JSON (nlohmann/json)
#include "json.hpp" 

using json = nlohmann::json;

// --- Helper para errores CUDA ---
template <typename T>
void cudaCheck(T result, const char* call, const char* file, int line) {
    if (result != hipSuccess) {
        std::cerr << "[CUDA ERROR] " << hipGetErrorString(result)
                  << " in " << call << " at " << file << ":" << line << std::endl;
        std::exit(EXIT_FAILURE);
    }
}
#define CUDA_CHECK(x) cudaCheck((x), #x, __FILE__, __LINE__)

// --- Constantes Generales ---
enum { MD5_DIGEST_LENGTH = 16, MAX_JSON_LEN = 65536, MAX_ULL_STR = 32 };

// --- IMPLEMENTACIÓN DE MD5 EN CUDA (DEVICE) ---
// Adaptada de varios ejemplos de dominio público. No es altamente optimizada
// para producción, pero es determinista y correcta para el propósito.

typedef unsigned int uint32;

// Funciones F, G, H, I (según la especificación MD5)
#define F(x, y, z) (((x) & (y)) | ((~x) & (z)))
#define G(x, y, z) (((x) & (z)) | ((y) & (~z)))
#define H(x, y, z) ((x) ^ (y) ^ (z))
#define I(x, y, z) ((y) ^ ((x) | (~z)))



// Rotación izquierda
#define ROTATE_LEFT(x, n) (((x) << (n)) | ((x) >> (32 - (n))))

// Constantes de rotación para las 4 rondas de MD5
#define S11 7
#define S12 12
#define S13 17
#define S14 22

#define S21 5
#define S22 9
#define S23 14
#define S24 20

#define S31 4
#define S32 11
#define S33 16
#define S34 23

#define S41 6
#define S42 10
#define S43 15
#define S44 21

// Operaciones para cada ronda (utilizan las Sxx definidas arriba)
#define FF(a, b, c, d, x, s, ac) { \
  (a) += F((b), (c), (d)) + (x) + (uint32)(ac); \
  (a) = ROTATE_LEFT((a), (s)); \
  (a) += (b); \
}
#define GG(a, b, c, d, x, s, ac) { \
  (a) += G((b), (c), (d)) + (x) + (uint32)(ac); \
  (a) = ROTATE_LEFT((a), (s)); \
  (a) += (b); \
}
#define HH(a, b, c, d, x, s, ac) { \
  (a) += H((b), (c), (d)) + (x) + (uint32)(ac); \
  (a) = ROTATE_LEFT((a), (s)); \
  (a) += (b); \
}
#define II(a, b, c, d, x, s, ac) { \
  (a) += I((b), (c), (d)) + (x) + (uint32)(ac); \
  (a) = ROTATE_LEFT((a), (s)); \
  (a) += (b); \
}

// Convertir 4 bytes a un uint32 (little-endian)
__device__ static uint32 bytes_to_uint32(const unsigned char* bytes) {
    return ((uint32)bytes[0]       ) |
           ((uint32)bytes[1] <<  8) |
           ((uint32)bytes[2] << 16) |
           ((uint32)bytes[3] << 24);
}

// Convertir uint32 a 4 bytes (little-endian)
__device__ static void uint32_to_bytes(uint32 val, unsigned char* bytes) {
    bytes[0] = (unsigned char)(val & 0xFF);
    bytes[1] = (unsigned char)((val >> 8) & 0xFF);
    bytes[2] = (unsigned char)((val >> 16) & 0xFF);
    bytes[3] = (unsigned char)((val >> 24) & 0xFF);
}

// Función MD5 real para CUDA device code
__device__ void md5_cuda_device(const unsigned char* data, size_t len, unsigned char* digest) {
    uint32 a_init = 0x67452301;
    uint32 b_init = 0xEFCDAB89;
    uint32 c_init = 0x98BADCFE;
    uint32 d_init = 0x10325476;

    uint32 a = a_init;
    uint32 b = b_init;
    uint32 c = c_init;
    uint32 d = d_init;

    // Calcular longitud de padding
    size_t original_len_bits = len * 8;
    // Añadir bit 1 de padding, luego ceros, y 64 bits para la longitud.
    // len + 1 (para 0x80) + 8 (para longitud de 64 bits)
    // El 63 es para redondear al siguiente múltiplo de 64
    size_t padded_len = (len + 1 + 8 + 63) / 64 * 64; 
    
    // Usar puntero dinámico o __shared__ si MAX_JSON_LEN + 64 es demasiado grande para la pila de hilo.
    // Para simplificar, asumimos que stack suficientemente grande o que el compilador lo optimiza.
    unsigned char padded_data[MAX_JSON_LEN + 64]; 
    
    // Copiar los datos originales
    for (size_t i = 0; i < len; ++i) {
        padded_data[i] = data[i];
    }
    // Añadir el bit 0x80 y rellenar con ceros
    padded_data[len] = 0x80; 
    for (size_t i = len + 1; i < padded_len - 8; ++i) {
        padded_data[i] = 0x00; 
    }
    // Añadir la longitud original en bits (64 bits, little-endian)
    uint32_to_bytes((uint32)(original_len_bits & 0xFFFFFFFF), &padded_data[padded_len - 8]);
    uint32_to_bytes((uint32)(original_len_bits >> 32), &padded_data[padded_len - 4]);

    for (size_t i = 0; i < padded_len; i += 64) {
        uint32 M[16];
        for (int j = 0; j < 16; ++j) {
            M[j] = bytes_to_uint32(&padded_data[i + j * 4]);
        }

        uint32 AA = a, BB = b, CC = c, DD = d;

        // Ronda 1
        FF(a, b, c, d, M[0], S11, 0xD76AA478);
        FF(d, a, b, c, M[1], S12, 0xE8C7B756);
        FF(c, d, a, b, M[2], S13, 0x242070DB);
        FF(b, c, d, a, M[3], S14, 0xC1BDCEEE);
        FF(a, b, c, d, M[4], S11, 0xF57C0FAF);
        FF(d, a, b, c, M[5], S12, 0x4787C62A);
        FF(c, d, a, b, M[6], S13, 0xA8304613);
        FF(b, c, d, a, M[7], S14, 0xFD469501);
        FF(a, b, c, d, M[8], S11, 0x698098D8);
        FF(d, a, b, c, M[9], S12, 0x8B44F7AF);
        FF(c, d, a, b, M[10], S13, 0xFFFF5BB1);
        FF(b, c, d, a, M[11], S14, 0x895CD7BE);
        FF(a, b, c, d, M[12], S11, 0x6B901122);
        FF(d, a, b, c, M[13], S12, 0xFD987193);
        FF(c, d, a, b, M[14], S13, 0xA679438E);
        FF(b, c, d, a, M[15], S14, 0x49B40821);

        // Ronda 2
        GG(a, b, c, d, M[1], S21, 0xF61E2562);
        GG(d, a, b, c, M[6], S22, 0xC040B340);
        GG(c, d, a, b, M[11], S23, 0x265E5A51);
        GG(b, c, d, a, M[0], S24, 0xE9B6C7AA);
        GG(a, b, c, d, M[5], S21, 0xD62F105D);
        GG(d, a, b, c, M[10], S22, 0x02441453);
        GG(c, d, a, b, M[15], S23, 0xD8A1E681);
        GG(b, c, d, a, M[4], S24, 0xE7D3FBC8);
        GG(a, b, c, d, M[9], S21, 0x21E1CDE6);
        GG(d, a, b, c, M[14], S22, 0xC33707D6);
        GG(c, d, a, b, M[3], S23, 0xF4D50D87);
        GG(b, c, d, a, M[8], S24, 0x455A14ED);
        GG(a, b, c, d, M[13], S21, 0xA9E3E905);
        GG(d, a, b, c, M[2], S22, 0xFCEFA3F8);
        GG(c, d, a, b, M[7], S23, 0x676F02D9);
        GG(b, c, d, a, M[12], S24, 0x8D2A4C8A);

        // Ronda 3
        HH(a, b, c, d, M[5], S31, 0xFFFA3942);
        HH(d, a, b, c, M[8], S32, 0x8771F681);
        HH(c, d, a, b, M[11], S33, 0x6D9D6122);
        HH(b, c, d, a, M[14], S34, 0xFDE5380C);
        HH(a, b, c, d, M[1], S31, 0xA4BEEA44);
        HH(d, a, b, c, M[4], S32, 0x4BDECFA9);
        HH(c, d, a, b, M[7], S33, 0xF6BB4B60);
        HH(b, c, d, a, M[10], S34, 0xBEBFBC70);
        HH(a, b, c, d, M[13], S31, 0x289B7EC6);
        HH(d, a, b, c, M[0], S32, 0xEAA127FA);
        HH(c, d, a, b, M[3], S33, 0xD4EF3085);
        HH(b, c, d, a, M[6], S34, 0x04881D05);
        HH(a, b, c, d, M[9], S31, 0xD9D4D039);
        HH(d, a, b, c, M[12], S32, 0xE6DB99E5);
        HH(c, d, a, b, M[15], S33, 0x1FA27CF8);
        HH(b, c, d, a, M[2], S34, 0xC4AC5665);

        // Ronda 4
        II(a, b, c, d, M[0], S41, 0xF4292244);
        II(d, a, b, c, M[7], S42, 0x432AFF97);
        II(c, d, a, b, M[14], S43, 0xAB9423A7);
        II(b, c, d, a, M[5], S44, 0xFC93A039);
        II(a, b, c, d, M[12], S41, 0x655B59C3);
        II(d, a, b, c, M[3], S42, 0x8F0CCC92);
        II(c, d, a, b, M[10], S43, 0xFFEFF47D);
        II(b, c, d, a, M[1], S44, 0x85845DD1);
        II(a, b, c, d, M[8], S41, 0x6FA87E4F);
        II(d, a, b, c, M[15], S42, 0xFE2CE6E0);
        II(c, d, a, b, M[6], S43, 0xA3014314);
        II(b, c, d, a, M[13], S44, 0x4E0811A1);
        II(a, b, c, d, M[4], S41, 0xF7537E82);
        II(d, a, b, c, M[11], S42, 0xBD3AF235);
        II(c, d, a, b, M[2], S43, 0x2AD7D2BB);
        II(b, c, d, a, M[9], S44, 0xEB86D391);

        // Sumar los resultados a los buffers iniciales
        a += AA; b += BB; c += CC; d += DD;
    }
    
    // Almacenar el digest final en el array de salida
    uint32_to_bytes(a, &digest[0]);
    uint32_to_bytes(b, &digest[4]);
    uint32_to_bytes(c, &digest[8]);
    uint32_to_bytes(d, &digest[12]);
}

// --- Convierte unsigned long long a string en device ---
__device__ void ulong_to_str(unsigned long long n, char* str, int& len) {
    len = 0;
    if (n == 0) {
        str[len++] = '0';
        str[len]   = '\0';
        return;
    }
    char tmp[MAX_ULL_STR];
    int  i = 0;
    unsigned long long temp_n = n; 
    while (temp_n > 0 && i < MAX_ULL_STR -1 ) { // Asegurar que no se desborde tmp
        tmp[i++] = char('0' + (temp_n % 10));
        temp_n /= 10;
    }
    // Invertir el string
    while (i > 0 && len < MAX_ULL_STR -1) { // Asegurar que no se desborde str
        str[len++] = tmp[--i];
    }
    str[len] = '\0'; 
}

// --- Estructura de resultados en GPU ---
struct GpuResult {
    unsigned long long found_nonce;
    unsigned char      block_hash[MD5_DIGEST_LENGTH];
    int                solution_found;
};

// --- Kernel de minería ---
__global__ void mineKernel(
    const char* prefix,      int prefix_len,
    const char* suffix,      int suffix_len,
    unsigned long long       start_nonce,
    unsigned long long       end_nonce,
    const unsigned char* target_prefix,
    int                      target_len,
    GpuResult* res_d
) {
    unsigned long long idx   = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned long long nonce = start_nonce + idx;

    // Primer punto de depuración: solo el primer hilo del primer bloque
    // Esto verifica que el kernel se lanza y los argumentos iniciales son correctos.
    if (threadIdx.x == 0 && blockIdx.x == 0 && nonce == start_nonce) { 
        printf("DEBUG GPU: Kernel alcanzado por Nonce %llu\n", nonce); 
    }

    if (nonce > end_nonce) return;
    
    // Si otro hilo ya encontró una solución, este hilo puede terminar temprano.
    if (atomicAdd(&res_d->solution_found, 0) > 0) return; 

    char buffer[MAX_JSON_LEN]; 
    int  pos = 0;

    // Copiar la parte del JSON antes del nonce (incluye "nonce":)
    if (pos + prefix_len >= MAX_JSON_LEN) return; 
    for (int i = 0; i < prefix_len; ++i) buffer[pos++] = prefix[i];

    // Convertir y copiar el nonce como string
    char nstr[MAX_ULL_STR];
    int  nlen;
    ulong_to_str(nonce, nstr, nlen);
    if (pos + nlen >= MAX_JSON_LEN) return; 
    for (int i = 0; i < nlen; ++i) buffer[pos++] = nstr[i];

    // Copiar la parte del JSON después del nonce (el resto de la cadena JSON)
    if (pos + suffix_len >= MAX_JSON_LEN) return; 
    for (int i = 0; i < suffix_len; ++i) buffer[pos++] = suffix[i];

    // Validación final del tamaño
    if (pos >= MAX_JSON_LEN) {
        return; 
    }

    // Segundo punto de depuración: muestra la cadena JSON completa para los primeros nonces
    // Limita la salida para que no sature la consola si el rango es grande
    if (threadIdx.x == 0 && blockIdx.x == 0 && nonce < 100) { 
        printf("GPU (Nonce %llu) Input (len %d): %.*s\n", nonce, pos, pos, buffer);
    }

    unsigned char hash[MD5_DIGEST_LENGTH];
    md5_cuda_device(reinterpret_cast<const unsigned char*>(buffer), pos, hash);

    bool ok = true;
    for (int i = 0; i < target_len; ++i) {
        if (hash[i] != target_prefix[i]) { ok = false; break; }
    }
    if (!ok) return;

    unsigned long long prev_nonce_in_result = atomicCAS(&res_d->found_nonce, ULLONG_MAX, nonce);
    if (prev_nonce_in_result == ULLONG_MAX) { 
        atomicExch(&res_d->solution_found, 1); 
        for (int i = 0; i < MD5_DIGEST_LENGTH; ++i) {
            res_d->block_hash[i] = hash[i];
        }
    }
}

// --- Funciones de Ayuda para el Host (CPU) ---
void log_host(const std::string& msg) {
    std::cerr << "[LOG] " << msg << std::endl;
}

unsigned long long parse_ull(const char* s, const std::string& name) {
    try {
        return std::stoull(s);
    } catch (const std::out_of_range& e) {
        log_host("Error parseando " + name + ": El valor \"" + s + "\" está fuera del rango de unsigned long long. Detalle: " + e.what());
        std::exit(1);
    } catch (const std::invalid_argument& e) {
        log_host("Error parseando " + name + ": El valor \"" + s + "\" no es un número válido. Detalle: " + e.what());
        std::exit(1);
    } catch (...) {
        log_host("Error desconocido parseando " + name + ": \"" + s + "\"");
        std::exit(1);
    }
}

json load_json(const std::string& path) {
    std::ifstream in(path);
    if (!in) {
        log_host("Error: No se pudo abrir el archivo JSON: " + path);
        std::exit(1);
    }
    std::stringstream ss;
    ss << in.rdbuf();
    try {
        return json::parse(ss.str());
    } catch (const json::parse_error& e) {
        log_host("Error de parseo JSON en '" + path + "': " + std::string(e.what()));
        std::exit(1);
    }
}

std::string extract_diff(const json& j) {
    if (j.count("configuracion") && j["configuracion"].count("dificultad")) {
        if (j["configuracion"]["dificultad"].is_string()) {
            return j["configuracion"]["dificultad"].get<std::string>();
        } else {
            log_host("Error: El campo 'dificultad' dentro de 'configuracion' no es una cadena.");
            std::exit(1);
        }
    }
    if (j.count("dificultad")) {
        if (j["dificultad"].is_string()) {
            return j["dificultad"].get<std::string>();
        } else {
            log_host("Error: El campo 'dificultad' no es una cadena.");
            std::exit(1);
        }
    }
    log_host("Error: Campo 'dificultad' no encontrado en el JSON. Asegúrate de que exista en 'configuracion' o directamente en la raíz.");
    std::exit(1);
}

int hex2bytes(const std::string& hex, unsigned char* out) {
    if (hex.length() % 2 != 0) {
        log_host("Error: La cadena hexadecimal de dificultad tiene longitud impar (" + std::to_string(hex.length()) + "). Debe ser par.");
        std::exit(1);
    }
    int bl = (int)hex.size()/2;
    for (int i = 0; i < bl; ++i) {
        try {
            out[i] = (unsigned char)std::stoi(hex.substr(2*i,2), nullptr, 16);
        } catch (const std::exception& e) {
            log_host("Error al convertir hex a bytes en '" + hex.substr(2*i,2) + "': " + e.what());
            std::exit(1);
        }
    }
    return bl;
}

// --- Función Principal (Main) del Host ---
int main(int argc, char** argv) {
    std::cerr.sync_with_stdio(true);
    std::cerr << std::unitbuf; 

    // Declaración de variables para el ámbito completo de main
    std::string prefix_str; 
    std::string suffix_str; 

    if (argc != 4) {
        log_host("Uso: MineroMD5CUDA <json_file> <start_nonce> <end_nonce>");
        return 1;
    }
    
    int dev;            
    hipDeviceProp_t prop; 

    try {
        log_host("Inicializando ejecución CUDA...");

        CUDA_CHECK(hipGetDevice(&dev));
        CUDA_CHECK(hipGetDeviceProperties(&prop, dev));
        log_host(std::string("GPU: ") + prop.name);

        json jb = load_json(argv[1]);
        std::string diff = extract_diff(jb);
        log_host("Dificultad: " + diff);

        unsigned char target[MD5_DIGEST_LENGTH];
        int           tlen = hex2bytes(diff, target);

        // 1. Eliminar el nonce si existe del JSON cargado
        if (jb.count("nonce")) {
            jb.erase("nonce");
        }

        // 2. Crear un nuevo JSON con el orden deseado para el hashing
        // Esto es crucial para que el nonce se inserte en la posición correcta.
        // Copiamos los campos en el orden que queremos para el hash
        json ordered_json;
        if (jb.count("index"))       ordered_json["index"] = jb["index"];
        ordered_json["nonce"] = 0; // Marcador temporal para el nonce
        if (jb.count("dificultad"))  ordered_json["dificultad"] = jb["dificultad"];
        if (jb.count("prev_hash"))   ordered_json["prev_hash"] = jb["prev_hash"];
        if (jb.count("timestamp"))   ordered_json["timestamp"] = jb["timestamp"];
        if (jb.count("transacciones")) ordered_json["transacciones"] = jb["transacciones"];
        // ¡Importante! Si tienes otros campos en tu JSON que no son 'index', 'nonce', 'dificultad',
        // 'prev_hash', 'timestamp' o 'transacciones', deberás añadirlos aquí en el orden correcto
        // para que sean parte del hash. Por ejemplo:
        // if (jb.count("otro_campo")) ordered_json["otro_campo"] = jb["otro_campo"];

        // Ahora, serializamos este JSON ordenado a una cadena compacta
        std::string compact_json_with_placeholder = ordered_json.dump();

        // 3. Encontrar la posición del marcador "nonce":0
        std::string nonce_placeholder = "\"nonce\":0";
        size_t nonce_pos_start = compact_json_with_placeholder.find(nonce_placeholder);

        if (nonce_pos_start == std::string::npos) {
            log_host("Error interno: No se encontró el marcador de nonce en el JSON reconstruido.");
            return 1;
        }

        // 4. Construir prefix_str y suffix_str
        // prefix_str será todo ANTES del "0" del nonce (incluye "nonce":)
        prefix_str = compact_json_with_placeholder.substr(0, nonce_pos_start + std::string("\"nonce\":").length());
        
        // suffix_str será todo DESPUÉS del "0" del nonce, incluyendo el '}' final
        suffix_str = compact_json_with_placeholder.substr(nonce_pos_start + nonce_placeholder.length());

        log_host("Prefix String para GPU (muestra): " + prefix_str.substr(0, std::min(prefix_str.length(), static_cast<size_t>(200))));
        log_host("Suffix String para GPU (muestra): " + suffix_str.substr(0, std::min(suffix_str.length(), static_cast<size_t>(200))));
        
        auto start_nonce = parse_ull(argv[2], "start_nonce");
        auto end_nonce   = parse_ull(argv[3], "end_nonce");
        if (start_nonce > end_nonce) {
            log_host("Error: Rango inválido. 'start_nonce' debe ser menor o igual a 'end_nonce'.");
            return 1;
        }

        char             *d_pre, *d_suf;
        unsigned char    *d_tar;
        GpuResult        *d_res;

        CUDA_CHECK(hipMalloc(&d_pre, prefix_str.size()+1));
        CUDA_CHECK(hipMalloc(&d_suf, suffix_str.size()+1));
        CUDA_CHECK(hipMalloc(&d_tar, tlen));
        CUDA_CHECK(hipMalloc(&d_res, sizeof(GpuResult)));

        CUDA_CHECK(hipMemcpy(d_pre, prefix_str.c_str(), prefix_str.size()+1, hipMemcpyHostToDevice));
        CUDA_CHECK(hipMemcpy(d_suf, suffix_str.c_str(), suffix_str.size()+1, hipMemcpyHostToDevice));
        CUDA_CHECK(hipMemcpy(d_tar, target, tlen, hipMemcpyHostToDevice));

        GpuResult init_res{ ULLONG_MAX, {0}, 0 }; 
        CUDA_CHECK(hipMemcpy(d_res, &init_res, sizeof(init_res), hipMemcpyHostToDevice)); 

        unsigned long long total_nonces_to_check = end_nonce - start_nonce + 1;
        int                threads_per_block = 256;
        int                num_blocks  = static_cast<int>((total_nonces_to_check + threads_per_block - 1) / threads_per_block);
        
        if (num_blocks > prop.maxGridSize[0]) {
            log_host("Advertencia: El número de bloques calculado (" + std::to_string(num_blocks) + 
                     ") excede el máximo de la GPU (" + std::to_string(prop.maxGridSize[0]) + "). Limitando bloques.");
            num_blocks = prop.maxGridSize[0];
        }

        log_host("Lanzando kernel con " + std::to_string(num_blocks) + " bloques de " + std::to_string(threads_per_block) + " hilos.");

        auto t0 = std::chrono::high_resolution_clock::now();
        mineKernel<<<num_blocks, threads_per_block>>>(
            d_pre, (int)prefix_str.size(),
            d_suf, (int)suffix_str.size(),
            start_nonce, end_nonce,
            d_tar, tlen,
            d_res
        );
        CUDA_CHECK(hipDeviceSynchronize()); 
        auto t1 = std::chrono::high_resolution_clock::now();

        GpuResult final_output;
        CUDA_CHECK(hipMemcpy(&final_output, d_res, sizeof(final_output), hipMemcpyDeviceToHost));
        
        hipFree(d_pre); 
        hipFree(d_suf); 
        hipFree(d_tar); 
        hipFree(d_res);

        double elapsed_ms = std::chrono::duration<double,std::milli>(t1 - t0).count();
        json   result_json_output;
        result_json_output["elapsed_time_ms"] = int64_t(elapsed_ms);

        if (final_output.solution_found) {
            char hash_str[33]; 
            for (int i = 0; i < MD5_DIGEST_LENGTH; ++i) {
                sprintf(hash_str + 2*i, "%02x", final_output.block_hash[i]);
            }
            hash_str[32] = '\0'; 

            result_json_output["status"]             = "solution_found";
            result_json_output["nonce_found"]        = final_output.found_nonce;
            result_json_output["block_hash_result"]  = std::string(hash_str);
            log_host("¡SOLUCIÓN ENCONTRADA! Nonce: " + std::to_string(final_output.found_nonce) + ", Hash: " + std::string(hash_str));
        } else {
            result_json_output["status"] = "no_solution_found";
            result_json_output["reason"] = "No se encontró un nonce válido en el rango especificado.";
            log_host("No se encontró solución en el rango especificado.");
        }

        std::cout << result_json_output.dump() << std::endl;
        return 0;

    } catch (const std::exception& e) {
        log_host(std::string("Error fatal inesperado: ") + e.what());
        return 1;
    }
}