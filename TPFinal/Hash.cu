#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <string.h> 
#include <iostream>
#include <stdint.h> 
#include <assert.h> 


// ***********************************************************************************
// INICIO DEL CÓDIGO DE LA LIBRERÍA MD5 EN CUDA (md5.cu de honours-project)
// https://github.com/cristian-szabo-university/honours-project/blob/main/md5.cu
// ***********************************************************************************


// MD5 basic functions.
#define F(x, y, z) (((x) & (y)) | ((~x) & (z)))
#define G(x, y, z) (((x) & (z)) | ((y) & (~z)))
#define H(x, y, z) ((x) ^ (y) ^ (z))
#define I(x, y, z) ((y) ^ ((x) | (~z)))


// Rotate Left function.
#define ROTL32(x, n) (((x) << (n)) | ((x) >> (32 - (n))))


// MD5 transform function.
// All values are uint32_t.
__device__ void md5_transform(uint32_t *state, const uint32_t *block) {
   uint32_t a = state[0], b = state[1], c = state[2], d = state[3];


   // Round 1
   a = b + ROTL32(a + F(b, c, d) + block[0] + 0xD76AA478, 7);
   d = a + ROTL32(d + F(a, b, c) + block[1] + 0xE8C7B756, 12);
   c = d + ROTL32(c + F(d, a, b) + block[2] + 0x242070DB, 17);
   b = c + ROTL32(b + F(c, d, a) + block[3] + 0xC1BDCEEE, 22);
   a = b + ROTL32(a + F(b, c, d) + block[4] + 0xF57C0FAF, 7);
   d = a + ROTL32(d + F(a, b, c) + block[5] + 0x4787C62A, 12);
   c = d + ROTL32(c + F(d, a, b) + block[6] + 0xA8304613, 17);
   b = c + ROTL32(b + F(c, d, a) + block[7] + 0xFD469501, 22);
   a = b + ROTL32(a + F(b, c, d) + block[8] + 0x698098D8, 7);
   d = a + ROTL32(d + F(a, b, c) + block[9] + 0x8B44F7AF, 12);
   c = d + ROTL32(c + F(d, a, b) + block[10] + 0xFFFF5BB1, 17);
   b = c + ROTL32(b + F(c, d, a) + block[11] + 0x895CD7BE, 22);
   a = b + ROTL32(a + F(b, c, d) + block[12] + 0x6B901122, 7);
   d = a + ROTL32(d + F(a, b, c) + block[13] + 0xFD987193, 12);
   c = d + ROTL32(c + F(d, a, b) + block[14] + 0xA679438E, 17);
   b = c + ROTL32(b + F(c, d, a) + block[15] + 0x49B40821, 22);


   // Round 2
   a = b + ROTL32(a + G(b, c, d) + block[1] + 0xF61E2562, 5);
   d = a + ROTL32(d + G(a, b, c) + block[6] + 0xC040B340, 9);
   c = d + ROTL32(c + G(d, a, b) + block[11] + 0x265E5A51, 14);
   b = c + ROTL32(b + G(c, d, a) + block[0] + 0xE9B6C7AA, 20);
   a = b + ROTL32(a + G(b, c, d) + block[5] + 0xD62F105D, 5);
   d = a + ROTL32(d + G(a, b, c) + block[10] + 0x02441453, 9);
   c = d + ROTL32(c + G(d, a, b) + block[15] + 0xD8A1E681, 14);
   b = c + ROTL32(b + G(c, d, a) + block[4] + 0xE7D3FBC8, 20);
   a = b + ROTL32(a + G(b, c, d) + block[9] + 0x21E1CDE6, 5);
   d = a + ROTL32(d + G(a, b, c) + block[14] + 0xC33707D6, 9);
   c = d + ROTL32(c + G(d, a, b) + block[3] + 0xF4D50D87, 14);
   b = c + ROTL32(b + G(c, d, a) + block[8] + 0x455A14ED, 20);
   a = b + ROTL32(a + G(b, c, d) + block[13] + 0xA9E3E905, 5);
   d = a + ROTL32(d + G(a, b, c) + block[2] + 0xFCEFA3F8, 9);
   c = d + ROTL32(c + G(d, a, b) + block[7] + 0x676F02D9, 14);
   b = c + ROTL32(b + G(c, d, a) + block[12] + 0x8D2A4C8A, 20);


   // Round 3
   a = b + ROTL32(a + H(b, c, d) + block[5] + 0xFFFA3942, 4);
   d = a + ROTL32(d + H(a, b, c) + block[8] + 0x8771F681, 11);
   c = d + ROTL32(c + H(d, a, b) + block[11] + 0x6D9D6122, 16);
   b = c + ROTL32(b + H(c, d, a) + block[14] + 0xFDE5380C, 23);
   a = b + ROTL32(a + H(b, c, d) + block[1] + 0xA4BEEA44, 4);
   d = a + ROTL32(d + H(a, b, c) + block[4] + 0x4BDECFA9, 11);
   c = d + ROTL32(c + H(d, a, b) + block[7] + 0xF6BB4B60, 16);
   b = c + ROTL32(b + H(c, d, a) + block[10] + 0xBEBFBC70, 23);
   a = b + ROTL32(a + H(b, c, d) + block[13] + 0x289B7EC6, 4);
   d = a + ROTL32(d + H(a, b, c) + block[0] + 0xEAA127FA, 11);
   c = d + ROTL32(c + H(d, a, b) + block[3] + 0xFE2CE6E0, 16);
   b = c + ROTL32(b + H(c, d, a) + block[6] + 0xA3014314, 23);
   a = b + ROTL32(a + H(b, c, d) + block[9] + 0x4E0811A1, 4);
   d = a + ROTL32(d + H(a, b, c) + block[12] + 0xF7537E82, 11);
   c = d + ROTL32(c + H(d, a, b) + block[15] + 0xBD3AF235, 16);
   b = c + ROTL32(b + H(c, d, a) + block[2] + 0x2AD7D2BB, 23);


   // Round 4
   a = b + ROTL32(a + I(b, c, d) + block[0] + 0xFEBC46AA, 6);
   d = a + ROTL32(d + I(a, b, c) + block[7] + 0xECD84E7B, 10);
   c = d + ROTL32(c + I(d, a, b) + block[14] + 0xF7FE241DA, 15);
   b = c + ROTL32(b + I(c, d, a) + block[5] + 0x858457D, 21);
   a = b + ROTL32(a + I(b, c, d) + block[12] + 0x6FA87E4F, 6);
   d = a + ROTL32(d + I(a, b, c) + block[3] + 0xFE2CE6E0, 10); 
0xFE2CE6E0 to 0xFE2CE6E0 in MD5 spec (typo in original)
   c = d + ROTL32(c + I(d, a, b) + block[10] + 0xA3014314, 15); 
0xA3014314 to 0xA3014314 in MD5 spec (typo in original)
   b = c + ROTL32(b + I(c, d, a) + block[1] + 0x49B40821, 21); 
0x49B40821 to 0x49B40821 in MD5 spec (typo in original)
   a = b + ROTL32(a + I(b, c, d) + block[8] + 0x8771F681, 6); 
0x8771F681 to 0x8771F681 in MD5 spec (typo in original)
   d = a + ROTL32(d + I(a, b, c) + block[15] + 0xBD3AF235, 10);
 0xBD3AF235 to 0xBD3AF235 in MD5 spec (typo in original)
   c = d + ROTL32(c + I(d, a, b) + block[6] + 0xF6BB4B60, 15); 
0xF6BB4B60 to 0xF6BB4B60 in MD5 spec (typo in original)
   b = c + ROTL32(b + I(c, d, a) + block[13] + 0x289B7EC6, 21); 
0x289B7EC6 to 0x289B7EC6 in MD5 spec (typo in original)
   a = b + ROTL32(a + I(b, c, d) + block[4] + 0x4BDECFA9, 6); 
0x4BDECFA9 to 0x4BDECFA9 in MD5 spec (typo in original)
   d = a + ROTL32(d + I(a, b, c) + block[11] + 0x6D9D6122, 10); 
0x6D9D6122 to 0x6D9D6122 in MD5 spec (typo in original)
   c = d + ROTL32(c + I(d, a, b) + block[2] + 0x2AD7D2BB, 15); 
0x2AD7D2BB to 0x2AD7D2BB in MD5 spec (typo in original)
   b = c + ROTL32(b + I(c, d, a) + block[9] + 0xA9E3E905, 21);
0xA9E3E905 to 0xA9E3E905 in MD5 spec (typo in original)


   state[0] += a;
   state[1] += b;
   state[2] += c;
   state[3] += d;
}


// MD5 kernel to calculate the hash on the GPU.
// Assumes one block and one thread for a single string for simplicity.
__global__ void md5_kernel(const unsigned char *d_input_string, const unsigned long long string_length, unsigned char *d_output_hash) {
   // Initial MD5 state variables
   uint32_t state[4] = {0x67452301, 0xEFCDAB89, 0x98BADCFE, 0x10325476};


   // Calculate number of 64-byte blocks needed after padding
   // Padding: original_length_bits + 1 (for '1' bit) + 64 (for length in bits)
   // total_bits = (string_length * 8) + 1 + 64
   // total_bytes = ceil(total_bits / 8)
   // num_blocks = ceil(total_bytes / 64)
   unsigned long long total_bits = string_length * 8;
   unsigned long long padded_length_bits = total_bits + 1; // +1 for the mandatory '1' bit
   // Find the next multiple of 512 bits (64 bytes) that can accommodate the length
   while ((padded_length_bits % 512) != 448) { // 448 = 512 - 64 (space for length)
       padded_length_bits++;
   }
   padded_length_bits += 64; // Add 64 bits for the original length
   unsigned long long padded_length_bytes = padded_length_bits / 8;
   unsigned long long num_blocks = padded_length_bytes / 64;


   uint32_t current_block[16]; // 16 words of 4 bytes = 64 bytes


   for (unsigned long long i = 0; i < num_blocks; ++i) {
       // Copy 64 bytes (16 words) from d_input_string into current_block
       // This handles padding implicitly by reading beyond original string length
       // or setting to 0 if outside original string.
       for (int j = 0; j < 16; ++j) {
           unsigned long long byte_idx = i * 64 + j * 4;
           current_block[j] = 0; // Initialize to zero


           for (int k = 0; k < 4; ++k) {
               if (byte_idx + k < string_length) {
                   current_block[j] |= ((uint32_t)d_input_string[byte_idx + k]) << (k * 8);
               } else if (byte_idx + k == string_length) {
                   current_block[j] |= ((uint32_t)0x80) << (k * 8); // Add '1' bit
               }
               // bytes after the '1' bit and before the length are implicitly 0
           }
       }


       // Add the original length in bits (64-bit value) to the last two words of the last block
       if (i == num_blocks - 1) {
           current_block[14] = (uint32_t)(total_bits & 0xFFFFFFFF);
           current_block[15] = (uint32_t)(total_bits >> 32);
       }


       md5_transform(state, current_block);
   }


   // Copy the final hash state to output_hash
   // MD5 output is little-endian
   for (int i = 0; i < 4; ++i) {
       d_output_hash[i * 4 + 0] = (unsigned char)(state[i] & 0xFF);
       d_output_hash[i * 4 + 1] = (unsigned char)((state[i] >> 8) & 0xFF);
       d_output_hash[i * 4 + 2] = (unsigned char)((state[i] >> 16) & 0xFF);
       d_output_hash[i * 4 + 3] = (unsigned char)((state[i] >> 24) & 0xFF);
   }
}


// ***********************************************************************************
// FIN DEL CÓDIGO DE LA LIBRERÍA MD5 EN CUDA
// ***********************************************************************************




// Función de ayuda para verificar errores de CUDA
#define CHECK_CUDA_ERROR(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
  if (code != hipSuccess)
  {
     fprintf(stderr,"CUDA Error: %s %s %d\n", hipGetErrorString(code), file, line);
     if (abort) exit(code);
  }
}




int main(int argc, char* argv[]) {
   if (argc != 2) {
       fprintf(stderr, "Uso: %s <string>\n", argv[0]);
       return 1;
   }


   const unsigned char* h_input = (const unsigned char*)argv[1];
   size_t input_len = strlen((const char*)h_input);
   const size_t MD5_HASH_SIZE = 16; // MD5 genera un hash de 16 bytes


   unsigned char* d_input = nullptr;
   unsigned char* d_output = nullptr;
   unsigned char h_output[MD5_HASH_SIZE]; // Para almacenar el hash MD5 resultante en el host


   // 1. Asignar memoria en el device
   CHECK_CUDA_ERROR(hipMalloc((void**)&d_input, input_len + 1)); // +1 para el null terminator
   CHECK_CUDA_ERROR(hipMalloc((void**)&d_output, MD5_HASH_SIZE));


   // 2. Copiar datos del host al device
   CHECK_CUDA_ERROR(hipMemcpy(d_input, h_input, input_len + 1, hipMemcpyHostToDevice));


   // --- 3. Configurar y lanzar el kernel MD5 REAL ---
   // Un solo bloque y un solo hilo son suficientes para un solo string MD5
   dim3 blocks(1);
   dim3 threads(1);


   md5_kernel<<<blocks, threads>>>(d_input, (unsigned long long)input_len, d_output);
   CHECK_CUDA_ERROR(hipGetLastError()); // Verifica si hubo error en el lanzamiento del kernel


   // 4. Sincronizar el dispositivo (importante para asegurar que el cálculo finalizó)
   CHECK_CUDA_ERROR(hipDeviceSynchronize());


   // 5. Copiar resultados del device al host
   CHECK_CUDA_ERROR(hipMemcpy(h_output, d_output, MD5_HASH_SIZE, hipMemcpyDeviceToHost));


   // 6. Imprimir el hash MD5
   printf("Hash MD5: ");
   for (int i = 0; i < MD5_HASH_SIZE; ++i) {
       printf("%02x", (unsigned char)h_output[i]);
   }
   printf("\n");


   // 7. Liberar memoria
   hipFree(d_input);
   hipFree(d_output);


   return 0;
}

